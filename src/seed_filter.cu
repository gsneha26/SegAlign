#include "hip/hip_runtime.h"
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scan.h>
#include <thrust/unique.h>
#include <thrust/sort.h>
#include "cuda_utils.h"
#include "parameters.h"
#include "seed_filter.h"
#include "seed_filter_interface.h"
#include "store.h"
#include "store_gpu.h"

// Each segmentPair is 16B
// With 64MB for the HSPs array per 1GB GPU memory
// With higher GPU memory, the size just linearly increases

#define MAX_HITS_PER_GB 4194304

int MAX_SEEDS;
int MAX_HITS;

uint32_t seed_size;
int **d_sub_mat;
int xdrop;
int hspthresh;
bool noentropy;

char** d_query_seq;
char** d_query_rc_seq;
uint32_t query_length[BUFFER_DEPTH];

uint64_t** d_seed_offsets;

uint32_t** d_hit_num_array;
std::vector<thrust::device_vector<uint32_t> > d_hit_num_vec;

uint32_t** d_done;
std::vector<thrust::device_vector<uint32_t> > d_done_vec;

segmentPair** d_hsp;
std::vector<thrust::device_vector<segmentPair> > d_hsp_vec;

segmentPair** d_hsp_reduced;
std::vector<thrust::device_vector<segmentPair> > d_hsp_reduced_vec;

struct hspEqual{
    __host__ __device__
        bool operator()(segmentPair x, segmentPair y){
            return ( ( (x.ref_start - x.query_start) == (y.ref_start - y.query_start) ) &&  ( ( (x.ref_start >= y.ref_start) && ( (x.ref_start + x.len) <= (y.ref_start + y.len) )  ) || ( ( y.ref_start >= x.ref_start ) && ( (y.ref_start + y.len) <= (x.ref_start + x.len) ) ) ) );
    }
};

struct hspComp{
    __host__ __device__
        bool operator()(segmentPair x, segmentPair y){
            if((x.ref_start - x.query_start) < (y.ref_start - y.query_start))
                return true;
            else if((x.ref_start - x.query_start) == (y.ref_start - y.query_start)){
		    if(x.ref_start < y.ref_start)
                    	return true;
                    else if(x.ref_start == y.ref_start){
                    	if(x.len < y.len)
                            return true;
                    	else if(x.len == y.len){
                            if(x.score > y.score)
                                return true;
                            else
                                return false;
                    	}
                    	else
                            return false;
                    }
                    else
                        return false;
                }
                else
                    return false;
       }
};

struct hspCompLastz{
    __host__ __device__
        bool operator()(segmentPair x, segmentPair y){
            if(x.query_start < y.query_start)
                return true;
            else if(x.query_start == y.query_start){
		    if(x.ref_start < y.ref_start)
                    	return true;
                    else if(x.ref_start == y.ref_start){
                    	if(x.len < y.len)
                            return true;
                    	else if(x.len == y.len){
                            if(x.score > y.score)
                                return true;
                            else
                                return false;
                    	}
                    	else
                            return false;
                    }
                    else
                        return false;
                }
                else
                    return false;
       }
};

__global__
void compress_string_rev_comp (uint32_t len, char* src_seq, char* dst_seq, char* dst_seq_rc){ 
    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    for (uint32_t i = start; i < len; i += stride) {
        char ch = src_seq[i];
        char dst = X_NT;
        char dst_rc = X_NT;
        if (ch == 'A'){
            dst = A_NT;
            dst_rc = T_NT;
        }
        else if (ch == 'C'){ 
            dst = C_NT;
            dst_rc = G_NT;
        }
        else if (ch == 'G'){
            dst = G_NT;
            dst_rc = C_NT;
        }
        else if (ch == 'T'){
            dst = T_NT;
            dst_rc = A_NT;
        }
        else if ((ch == 'a') || (ch == 'c') || (ch == 'g') || (ch == 't')){
            dst = L_NT;
            dst_rc = L_NT;
        }
        else if ((ch == 'n') || (ch == 'N')){
            dst = N_NT;
            dst_rc = N_NT;
        }
        else if (ch == '&'){
            dst = E_NT;
            dst_rc = E_NT;
        }
        dst_seq[i] = dst;
        dst_seq_rc[len -1 -i] = dst_rc;
    }
}

__global__
void find_num_hits (int num_seeds, const uint32_t* __restrict__ d_index_table, uint64_t* seed_offsets, uint32_t* seed_hit_num){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    uint32_t num_seed_hit;
    uint32_t seed;
    
    for (uint32_t id = start; id < num_seeds; id += stride) {
        seed = (seed_offsets[id] >> 32);

        // start and end from the seed block_id table
        num_seed_hit = d_index_table[seed];
        if (seed > 0){
            num_seed_hit -= d_index_table[seed-1];
        }

        seed_hit_num[id] = num_seed_hit;
    }
}

__global__
void find_hits (const uint32_t* __restrict__  d_index_table, const uint32_t* __restrict__ d_pos_table, uint64_t*  d_seed_offsets, uint32_t seed_size, uint32_t* seed_hit_num, int num_hits, segmentPair* d_hsp, uint32_t start_seed_index, uint32_t start_hit_index){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int warp_size = warpSize;
    int lane_id = thread_id%warp_size;
    int warp_id = (thread_id-lane_id)/warp_size;

    __shared__ uint32_t start, end;
    __shared__ uint32_t seed;
    __shared__ uint64_t seed_offset;

    __shared__ uint32_t ref_loc[NUM_WARPS];
    __shared__ uint32_t query_loc;
    __shared__ uint32_t seed_hit_prefix;

    if(thread_id == 0){
        seed_offset = d_seed_offsets[block_id+start_seed_index];
        seed = (seed_offset >> 32);
        query_loc = ((seed_offset << 32) >> 32) + seed_size;

        // start and end from the seed block_id table
        end = d_index_table[seed];
        start = 0;
        if (seed > 0){
            start = d_index_table[seed-1];
        }
        seed_hit_prefix = seed_hit_num[block_id+start_seed_index]; 
    }
    __syncthreads();


    for (int id1 = start; id1 < end; id1 += NUM_WARPS) {
        if(id1+warp_id < end){ 
            if(lane_id == 0){ 
                ref_loc[warp_id]   = d_pos_table[id1+warp_id] + seed_size;
                int dram_address = seed_hit_prefix -id1 - warp_id+start-1-start_hit_index;

                d_hsp[dram_address].ref_start = ref_loc[warp_id];
                d_hsp[dram_address].query_start = query_loc; 
                d_hsp[dram_address].len = 0;
                d_hsp[dram_address].score = 0;
            }
        }
    }
}

__global__
void find_hsps (const char* __restrict__  d_ref_seq, const char* __restrict__  d_query_seq, uint32_t ref_len, uint32_t query_len, int *d_sub_mat, bool noentropy, int xdrop, int hspthresh, int num_hits, segmentPair* d_hsp, uint32_t* d_done){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int num_blocks = gridDim.x;
    int warp_size = warpSize;
    int lane_id = thread_id%warp_size;
    int warp_id = (thread_id-lane_id)/warp_size;

    __shared__ uint32_t ref_loc[NUM_WARPS];
    __shared__ uint32_t query_loc[NUM_WARPS];
    __shared__ int total_score[NUM_WARPS];
    __shared__ int prev_score[NUM_WARPS];
    __shared__ int prev_max_score[NUM_WARPS];
    __shared__ int prev_max_pos[NUM_WARPS];
    __shared__ bool edge_found[NUM_WARPS]; 
    __shared__ bool xdrop_found[NUM_WARPS]; 
    __shared__ bool new_max_found[NUM_WARPS]; 
    __shared__ uint32_t left_extent[NUM_WARPS];
    __shared__ int extent[NUM_WARPS];
    __shared__ uint32_t tile[NUM_WARPS];
    __shared__ double entropy[NUM_WARPS];

    int thread_score;
    int max_thread_score;
    int max_pos;
    int temp_pos;
    bool xdrop_done;
    bool temp_xdrop_done;
    int temp;
    short count[4];
    short count_del[4];
    char r_chr;
    char q_chr;
    uint32_t ref_pos;
    uint32_t query_pos;
    int pos_offset;

    __shared__ int sub_mat[NUC2];

    if(thread_id < NUC2){
        sub_mat[thread_id] = d_sub_mat[thread_id];
    }
    __syncthreads();

    for(int hid0 = block_id*NUM_WARPS; hid0 < num_hits; hid0 += NUM_WARPS*num_blocks){ 
        int hid = hid0 + warp_id; 

        if(hid < num_hits){
            if(lane_id == 0){
                ref_loc[warp_id] = d_hsp[hid].ref_start;
                query_loc[warp_id] = d_hsp[hid].query_start;
                total_score[warp_id] = 0; 
            }
        }
        else{
            if(lane_id == 0){

                ref_loc[warp_id] = d_hsp[hid0].ref_start;
                query_loc[warp_id] = d_hsp[hid0].query_start;
                total_score[warp_id] = 0; 
            }
        }
        __syncwarp();


        //////////////////////////////////////////////////////////////////
        //Right extension

        if(lane_id ==0){
            tile[warp_id] = 0;
            xdrop_found[warp_id] = false;
            edge_found[warp_id] = false;
            new_max_found[warp_id] = false;
            entropy[warp_id] = 1.0f;
            prev_score[warp_id] = 0;
            prev_max_score[warp_id] = 0;
            prev_max_pos[warp_id] = -1;
            extent[warp_id] = 0;
        }

        count[0] = 0;
        count[1] = 0;
        count[2] = 0;
        count[3] = 0;
        count_del[0] = 0;
        count_del[1] = 0;
        count_del[2] = 0;
        count_del[3] = 0;
        max_pos = 0;

        __syncwarp();

        while(!xdrop_found[warp_id] && !edge_found[warp_id]){
            pos_offset = lane_id + tile[warp_id];
            ref_pos   = ref_loc[warp_id] + pos_offset;
            query_pos = query_loc[warp_id] + pos_offset;
            thread_score = 0;

            if(ref_pos < ref_len && query_pos < query_len){
                r_chr = d_ref_seq[ref_pos];
                q_chr = d_query_seq[query_pos];
                thread_score = sub_mat[r_chr*NUC+q_chr];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                if(lane_id >= offset){
                    thread_score += temp;
                }
            }


            thread_score += prev_score[warp_id];
            if(thread_score > prev_max_score[warp_id]){
                max_thread_score = thread_score;
                max_pos = pos_offset;
            }
            else{
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }

            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }

            xdrop_done = ((max_thread_score-thread_score) > xdrop);
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp_xdrop_done = __shfl_up_sync(0xFFFFFFFF, xdrop_done, offset);

                if(lane_id >= offset){
                    xdrop_done |= temp_xdrop_done;
                }
            }

            if(xdrop_done == 1){
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }
            __syncwarp();

            if(lane_id == warp_size-1){

                if(max_pos > prev_max_pos[warp_id])
                    new_max_found[warp_id] = true;
                else
                    new_max_found[warp_id] = false;

                if(xdrop_done){
                    total_score[warp_id] += max_thread_score;
                    xdrop_found[warp_id] = true;
                    extent[warp_id] = max_pos;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else if(ref_pos >= ref_len || query_pos >= query_len){
                    total_score[warp_id] += max_thread_score;
                    edge_found[warp_id] = true;
                    extent[warp_id] = max_pos;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else{
                    prev_score[warp_id] = thread_score;
                    prev_max_score[warp_id] = max_thread_score;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id]+= warp_size;
                }
            }
            __syncwarp();

            if(new_max_found[warp_id]){
                for(int i = 0; i < 4; i++){
                    count[i] = count[i] + count_del[i];
                    count_del[i] = 0;
                }
            }
            __syncwarp();

            if(r_chr == q_chr){
                if(pos_offset <= prev_max_pos[warp_id]){
                    count[r_chr] += 1;
                }
                else{
                    count_del[r_chr] += 1;
                }
            }
            __syncwarp();
        }

        __syncwarp();

        ////////////////////////////////////////////////////////////////
        //Left extension

        if(lane_id ==0){
            tile[warp_id] = 0;
            xdrop_found[warp_id] = false;
            edge_found[warp_id] = false;
            new_max_found[warp_id] = false;
            prev_score[warp_id] = 0;
            prev_max_score[warp_id] = 0;
            prev_max_pos[warp_id] = 0;
            left_extent[warp_id] = 0;
        }

        count_del[0] = 0;
        count_del[1] = 0;
        count_del[2] = 0;
        count_del[3] = 0;
        max_pos = 0;
        __syncwarp();

        while(!xdrop_found[warp_id] && !edge_found[warp_id]){
            pos_offset = lane_id+1+tile[warp_id];
            thread_score = 0;

            if(ref_loc[warp_id] >= pos_offset  && query_loc[warp_id] >= pos_offset){
                ref_pos   = ref_loc[warp_id] - pos_offset;
                query_pos = query_loc[warp_id] - pos_offset;
                r_chr = d_ref_seq[ref_pos];
                q_chr = d_query_seq[query_pos];
                thread_score = sub_mat[r_chr*NUC+q_chr];
            }

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                if(lane_id >= offset){
                    thread_score += temp;
                }
            }

            thread_score += prev_score[warp_id];
            if(thread_score > prev_max_score[warp_id]){
                max_thread_score = thread_score;
                max_pos = pos_offset;
            }
            else{
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }

            xdrop_done = ((max_thread_score-thread_score) > xdrop);
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp_xdrop_done = __shfl_up_sync(0xFFFFFFFF, xdrop_done, offset);

                if(lane_id >= offset){
                    xdrop_done |= temp_xdrop_done;
                }
            }

            if(xdrop_done){
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }
            __syncwarp();

            if(lane_id == warp_size-1){

                if(max_pos > prev_max_pos[warp_id])
                    new_max_found[warp_id] = true;
                else
                    new_max_found[warp_id] = false;

                if(xdrop_done){
                    total_score[warp_id]+=max_thread_score;
                    xdrop_found[warp_id] = true;
                    left_extent[warp_id] = max_pos;
                    extent[warp_id] += left_extent[warp_id];
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else if(ref_loc[warp_id] < pos_offset || query_loc[warp_id] < pos_offset){
                    total_score[warp_id]+=max_thread_score;
                    edge_found[warp_id] = true;
                    left_extent[warp_id] = max_pos;
                    extent[warp_id] += left_extent[warp_id];
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else{
                    prev_score[warp_id] = thread_score;
                    prev_max_score[warp_id] = max_thread_score;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id]+=warp_size;
                }
            }
            __syncwarp();

            if(new_max_found[warp_id]){
                for(int i = 0; i < 4; i++){
                    count[i] = count[i] + count_del[i];
                    count_del[i] = 0;
                }
            }
            __syncwarp();

            if(r_chr == q_chr){
                if(pos_offset <= prev_max_pos[warp_id]){
                    count[r_chr] += 1;
                }
                else{
                    count_del[r_chr] += 1;
                }
            }
            __syncwarp();
        }

        //////////////////////////////////////////////////////////////////

        if(total_score[warp_id] >= hspthresh && total_score[warp_id] <= 3*hspthresh && !noentropy){
            for(int i = 0; i < 4; i++){
#pragma unroll
                for (int offset = 1; offset < warp_size; offset = offset << 1){
                    count[i] += __shfl_up_sync(0xFFFFFFFF, count[i], offset);
                }
            }
            __syncwarp();

            if(lane_id == warp_size-1 && ((count[0]+count[1]+count[2]+count[3]) >= 20)){

                entropy[warp_id] = 0.f;
                for(int i = 0; i < 4; i++){
                    entropy[warp_id] += ((double) count[i])/((double) (extent[warp_id]+1)) * ((count[i] != 0) ? log(((double) count[i]) / ((double) (extent[warp_id]+1))): 0.f); 
                }
                entropy[warp_id] = -entropy[warp_id]/log(4.0f);
            }
        }
        __syncwarp();

        //////////////////////////////////////////////////////////////////

        if(hid < num_hits){
            if(lane_id == 0){

                if( ((int) (((float) total_score[warp_id])  * entropy[warp_id])) >= hspthresh){
                    d_hsp[hid].ref_start = ref_loc[warp_id] - left_extent[warp_id];
                    d_hsp[hid].query_start = query_loc[warp_id] - left_extent[warp_id];
                    d_hsp[hid].len = extent[warp_id];
                    if(entropy[warp_id] > 0)
                        d_hsp[hid].score = total_score[warp_id]*entropy[warp_id];
                    d_done[hid] = 1;
                }
                else{
                    d_hsp[hid].ref_start = ref_loc[warp_id];
                    d_hsp[hid].query_start = query_loc[warp_id];
                    d_hsp[hid].len = 0;
                    d_hsp[hid].score = 0;
                    d_done[hid] = 0;
                }
            }
        }
        __syncwarp();
    }
}

__global__
void compress_output (uint32_t* d_done, segmentPair* d_hsp, segmentPair* d_hsp_reduced, int num_hits){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;
    int index = 0;

    for (uint32_t id = start; id < num_hits; id += stride) {
        index = d_done[id];

        if(id > 0){
            if(index > d_done[id-1]){
                d_hsp_reduced[index-1] = d_hsp[id];
            }
        }
        else{
            if(index == 1){
                d_hsp_reduced[0] = d_hsp[0];
            }
        }
    }
}

std::vector<segmentPair> SeedAndFilter (std::vector<uint64_t> seed_offset_vector, bool rev, uint32_t buffer){

    uint32_t num_hits = 0;
    uint32_t total_anchors = 0;

    uint32_t num_seeds = seed_offset_vector.size();
    if(num_seeds > MAX_SEEDS){
	    printf("MAX_SEEDS exceeded\n");
    }

    assert(num_seeds <= MAX_SEEDS);

    uint64_t* tmp_offset = (uint64_t*) malloc(num_seeds*sizeof(uint64_t));
    for (uint32_t i = 0; i < num_seeds; i++) {
        tmp_offset[i] = seed_offset_vector[i];
    }

    int g;
    std::unique_lock<std::mutex> locker(mu);
    if (available_gpus.empty()) {
        cv.wait(locker, [](){return !available_gpus.empty();});
    }
    g = available_gpus.back();
    available_gpus.pop_back();
    locker.unlock();

    check_cuda_setDevice(g, "SeedAndFilter");

    check_cuda_memcpy((void*)d_seed_offsets[g], (void*)tmp_offset, num_seeds*sizeof(uint64_t), hipMemcpyHostToDevice, "seed_offsets");

    find_num_hits <<<MAX_BLOCKS, MAX_THREADS>>> (num_seeds, d_index_table[g], d_seed_offsets[g], d_hit_num_array[g]);

    thrust::inclusive_scan(d_hit_num_vec[g].begin(), d_hit_num_vec[g].begin() + num_seeds, d_hit_num_vec[g].begin());

    check_cuda_memcpy((void*)&num_hits, (void*)(d_hit_num_array[g]+num_seeds-1), sizeof(uint32_t), hipMemcpyDeviceToHost, "num_hits");
    
    int num_iter;
    uint32_t iter_hit_limit;

    if(num_hits < MAX_HITS){
	    num_iter = 2;
	    iter_hit_limit = num_hits;
    }
    else{
	    num_iter = num_hits/MAX_HITS+2;
	    iter_hit_limit = MAX_HITS;
    }

    thrust::device_vector<uint32_t> limit_pos (num_iter); 

    for(int i = 0; i < num_iter-1; i++){
        thrust::device_vector<uint32_t>::iterator result_end = thrust::lower_bound(d_hit_num_vec[g].begin(), d_hit_num_vec[g].begin()+num_seeds, iter_hit_limit);
        uint32_t pos = thrust::distance(d_hit_num_vec[g].begin(), result_end)-1;
        limit_pos[i] = pos;
        iter_hit_limit = d_hit_num_vec[g][pos]+MAX_HITS;
	if(iter_hit_limit > num_hits)
		iter_hit_limit = num_hits;
    }

    limit_pos[num_iter-1] = num_seeds-1;

    if(limit_pos[num_iter-1] == limit_pos[num_iter-2]){
	    num_iter--;
    }

    segmentPair** h_hsp = (segmentPair**) malloc(num_iter*sizeof(segmentPair*));
    uint32_t* num_anchors = (uint32_t*) calloc(num_iter, sizeof(uint32_t));

    uint32_t start_seed_index = 0;
    uint32_t start_hit_val = 0;
    uint32_t iter_num_seeds, iter_num_hits;

    if(num_hits > 0){
        
        for(int i = 0; i < num_iter; i++){
            iter_num_seeds = limit_pos[i] + 1 - start_seed_index;
            iter_num_hits  = d_hit_num_vec[g][limit_pos[i]] - start_hit_val;

            find_hits <<<iter_num_seeds, BLOCK_SIZE>>> (d_index_table[g], d_pos_table[g], d_seed_offsets[g], seed_size, d_hit_num_array[g], iter_num_hits, d_hsp[g], start_seed_index, start_hit_val);

            if(rev){
                find_hsps <<<1024, BLOCK_SIZE>>> (d_ref_seq[g], d_query_rc_seq[buffer*NUM_DEVICES+g], ref_len, query_length[buffer], d_sub_mat[g], noentropy, xdrop, hspthresh, iter_num_hits, d_hsp[g], d_done[g]);
            }
            else{
                find_hsps <<<1024, BLOCK_SIZE>>> (d_ref_seq[g], d_query_seq[buffer*NUM_DEVICES+g], ref_len, query_length[buffer], d_sub_mat[g], noentropy, xdrop, hspthresh, iter_num_hits, d_hsp[g], d_done[g]);
            }

            thrust::inclusive_scan(d_done_vec[g].begin(), d_done_vec[g].begin() + iter_num_hits, d_done_vec[g].begin());

            check_cuda_memcpy((void*)&num_anchors[i], (void*)(d_done[g]+iter_num_hits-1), sizeof(uint32_t), hipMemcpyDeviceToHost, "num_anchors");

            if(num_anchors[i] > 0){
                compress_output <<<MAX_BLOCKS, MAX_THREADS>>>(d_done[g], d_hsp[g], d_hsp_reduced[g], iter_num_hits);

                thrust::stable_sort(d_hsp_reduced_vec[g].begin(), d_hsp_reduced_vec[g].begin()+num_anchors[i], hspComp());
                
                thrust::device_vector<segmentPair>::iterator result_end = thrust::unique_copy(d_hsp_reduced_vec[g].begin(), d_hsp_reduced_vec[g].begin()+num_anchors[i], d_hsp_vec[g].begin(),  hspEqual());

                num_anchors[i] = thrust::distance(d_hsp_vec[g].begin(), result_end), num_anchors[i];

		thrust::stable_sort(d_hsp_vec[g].begin(), d_hsp_vec[g].begin()+num_anchors[i], hspCompLastz());

                total_anchors += num_anchors[i];

                h_hsp[i] = (segmentPair*) calloc(num_anchors[i], sizeof(segmentPair));

                check_cuda_memcpy((void*)h_hsp[i], (void*)d_hsp[g], num_anchors[i]*sizeof(segmentPair), hipMemcpyDeviceToHost, "hsp_output");
            }

            start_seed_index = limit_pos[i] + 1;
            start_hit_val = d_hit_num_vec[g][limit_pos[i]];
        }
    }

    limit_pos.clear();

    {
        std::unique_lock<std::mutex> locker(mu);
        available_gpus.push_back(g);
        locker.unlock();
        cv.notify_one();
    }
    std::vector<segmentPair> gpu_filter_output;

    segmentPair first_el;
    first_el.len = total_anchors;
    first_el.score = num_hits;
    gpu_filter_output.push_back(first_el);

    if(total_anchors > 0){
        for(int it = 0; it < num_iter; it++){

            for(int i = 0; i < num_anchors[it]; i++){
                gpu_filter_output.push_back(h_hsp[it][i]);
            }

            if(num_anchors[it] > 0){
                free(h_hsp[it]);
            }
        }
    }
    
    free(h_hsp);
    free(num_anchors);
    free(tmp_offset);
    return gpu_filter_output;
}

void InitializeProcessor (bool transition, uint32_t WGA_CHUNK, uint32_t input_seed_size, int* sub_mat, int input_xdrop, int input_hspthresh, bool input_noentropy){

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    float global_mem_gb = static_cast<float>(deviceProp.totalGlobalMem / 1073741824.0f);

    if(transition)
        MAX_SEEDS = 13*WGA_CHUNK;
    else
        MAX_SEEDS = WGA_CHUNK;

    MAX_HITS = MAX_HITS_PER_GB*global_mem_gb;

    seed_size = input_seed_size;
    xdrop = input_xdrop;
    hspthresh = input_hspthresh;
    noentropy = input_noentropy;

    d_sub_mat = (int**) malloc(NUM_DEVICES*sizeof(int*));

    d_query_seq = (char**) malloc(BUFFER_DEPTH*NUM_DEVICES*sizeof(char*));
    d_query_rc_seq = (char**) malloc(BUFFER_DEPTH*NUM_DEVICES*sizeof(char*));

    d_seed_offsets = (uint64_t**) malloc(NUM_DEVICES*sizeof(uint64_t*));

    d_hit_num_array = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_hit_num_vec.reserve(NUM_DEVICES);

    d_done = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_done_vec.reserve(NUM_DEVICES);

    d_hsp = (segmentPair**) malloc(NUM_DEVICES*sizeof(segmentPair*));
    d_hsp_vec.reserve(NUM_DEVICES);

    d_hsp_reduced = (segmentPair**) malloc(NUM_DEVICES*sizeof(segmentPair*));
    d_hsp_reduced_vec.reserve(NUM_DEVICES);

    segmentPair zeroHsp;
    zeroHsp.ref_start = 0;
    zeroHsp.query_start = 0;
    zeroHsp.len = 0;
    zeroHsp.score = 0;

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "InitializeProcessor");

        check_cuda_malloc((void**)&d_sub_mat[g], NUC2*sizeof(int), "sub_mat"); 

        check_cuda_memcpy((void*)d_sub_mat[g], (void*)sub_mat, NUC2*sizeof(int), hipMemcpyHostToDevice, "sub_mat");

        check_cuda_malloc((void**)&d_seed_offsets[g], MAX_SEEDS*sizeof(uint64_t), "seed_offsets");

        d_hit_num_vec.emplace_back(MAX_SEEDS, 0);
        d_hit_num_array[g] = thrust::raw_pointer_cast(d_hit_num_vec.at(g).data());

        d_done_vec.emplace_back(MAX_HITS, 0);
        d_done[g] = thrust::raw_pointer_cast(d_done_vec.at(g).data());

        d_hsp_vec.emplace_back(MAX_HITS, zeroHsp);
        d_hsp[g] = thrust::raw_pointer_cast(d_hsp_vec.at(g).data());

        d_hsp_reduced_vec.emplace_back(MAX_HITS, zeroHsp);
        d_hsp_reduced[g] = thrust::raw_pointer_cast(d_hsp_reduced_vec.at(g).data());

        available_gpus.push_back(g);
    }
}

void SendQueryWriteRequest (size_t start_addr, uint32_t len, uint32_t buffer){

    query_length[buffer] = len;

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "SendQueryWriteRequest");

        char* d_query_seq_tmp;
        check_cuda_malloc((void**)&d_query_seq_tmp, len*sizeof(char), "tmp query_seq"); 

        check_cuda_memcpy((void*)d_query_seq_tmp, (void*)(query_DRAM->buffer + start_addr), len*sizeof(char), hipMemcpyHostToDevice, "query_seq");

        check_cuda_malloc((void**)&d_query_seq[buffer*NUM_DEVICES+g], len*sizeof(char), "query_seq"); 
        check_cuda_malloc((void**)&d_query_rc_seq[buffer*NUM_DEVICES+g], len*sizeof(char), "query_rc_seq"); 

        compress_string_rev_comp <<<MAX_BLOCKS, MAX_THREADS>>> (len, d_query_seq_tmp, d_query_seq[buffer*NUM_DEVICES+g], d_query_rc_seq[buffer*NUM_DEVICES+g]);

        check_cuda_free((void*)d_query_seq_tmp, "d_query_seq_tmp");
    }
}

void ClearQuery(uint32_t buffer){

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "ClearQuery");

        check_cuda_free((void*)d_query_seq[buffer*NUM_DEVICES+g], "d_query_seq");
        check_cuda_free((void*)d_query_rc_seq[buffer*NUM_DEVICES+g], "d_query_rc_seq");
    }
}

void ShutdownProcessor(){

    d_done_vec.clear();
    d_hit_num_vec.clear();
    d_hsp_vec.clear();
    d_hsp_reduced_vec.clear();

    hipDeviceReset();
}

InitializeProcessor_ptr g_InitializeProcessor = InitializeProcessor;
SendQueryWriteRequest_ptr g_SendQueryWriteRequest = SendQueryWriteRequest;
SeedAndFilter_ptr g_SeedAndFilter = SeedAndFilter;
ClearQuery_ptr g_ClearQuery = ClearQuery;
ShutdownProcessor_ptr g_ShutdownProcessor = ShutdownProcessor;
