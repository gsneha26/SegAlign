#include "hip/hip_runtime.h"
#include <thrust/scan.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "graph.h"
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <algorithm>
#include <mutex>
#include <cstring>
#include <cstdlib>
#include "tbb/scalable_allocator.h"
#include <stdio.h>
#include <stdlib.h>

#define MAX_BLOCKS 1<<16
#define MAX_THREADS 1024 
#define TILE_SIZE 64
#define NUM_TILES 2
#define BLOCK_SIZE 32

std::mutex gpu_lock;

const char A_NT = 0;
const char C_NT = 1;
const char G_NT = 2;
const char T_NT = 3;
const char N_NT = 4;

int mat_offset[] = {0, 1, 3, 6};                                                             
int *d_sub_mat;

int err;                            
int check_status = 0;

int ref_len;
int query_len;
int seed_size;

struct timeval time1, time2, time3, time4, time5; 
long useconds1, seconds1, mseconds1;

char* d_ref_seq;
char* d_query_seq;
uint64_t* d_seed_offsets;
uint32_t* d_index_table;
uint32_t* d_num_seed_hits;
uint64_t* d_pos_table;
uint32_t* d_r_starts;
uint32_t* d_q_starts;
uint32_t* d_len;
bool* d_done;
uint64_t* h_seed_offsets;
int *sub_mat;

__global__
void compress_string (uint32_t n, char* src_seq, char* dst_seq){ 
    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    uint32_t len = n;

    for (uint32_t i = start; i < len; i += stride) {
        char ch = src_seq[i];
        char dst = N_NT;
        if ((ch == 'a') || (ch == 'A'))
            dst = A_NT;
        else if ((ch == 'c') || (ch == 'C'))
            dst = C_NT;
        else if ((ch == 'g') || (ch == 'G'))
            dst = G_NT;
        else if ((ch == 't') || (ch == 'T'))
            dst = T_NT;
        dst_seq[i] = dst;
    }
}

__global__
void find_num_hits (int num_seeds, uint32_t* d_index_table, uint64_t* seed_offsets, uint32_t* d_num_seed_hits, int* seed_hit_num){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    uint32_t num_seed_hit;
    uint32_t seed;
    
    for (uint32_t id = start; id < num_seeds; id += stride) {
        seed = (seed_offsets[id] >> 32);

        // start and end from the seed block_id table
        num_seed_hit = d_index_table[seed];
        if (seed > 0){
            num_seed_hit -= d_index_table[seed-1];
        }

        d_num_seed_hits[id] = num_seed_hit;
        seed_hit_num[id] = num_seed_hit;
    }

}

__global__
void find_anchors3 (int num_seeds, char* d_ref_seq, char* d_query_seq, uint32_t* d_index_table, uint64_t* d_pos_table, uint64_t* d_seed_offsets, int *d_sub_mat, int xdrop, int xdrop_threshold, uint32_t* d_r_starts, uint32_t* d_q_starts, uint32_t* d_len, bool* d_done, int ref_len, int query_len, int seed_size, int* seed_hit_num, int num_hits){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int warp_size = warpSize;

    __shared__ uint32_t start, end;
    __shared__ uint32_t q_start;
    __shared__ uint32_t seed;
    __shared__ uint64_t seed_offset;

    __shared__ uint32_t ref_loc;
    __shared__ uint32_t query_loc;
    __shared__ int total_score;
    __shared__ int prev_score;
    __shared__ int prev_max_score;
    __shared__ bool right_edge; 
    __shared__ bool left_edge; 
    __shared__ bool right_xdrop_found; 
    __shared__ bool left_xdrop_found; 
    __shared__ uint32_t left_extent;
    __shared__ uint32_t right_extent;

    int thread_score;
    int max_thread_score;
    bool xdrop_done;
    int temp;
    int tile = 0;
    int ref_pos;
    int query_pos;

    __shared__ int sub_mat[25];

    if(thread_id < 25){
        sub_mat[thread_id] = d_sub_mat[thread_id];
    }

    if(thread_id == 0){
        seed_offset = d_seed_offsets[block_id];
    }
    __syncthreads();

    seed = (seed_offset >> 32);
    q_start = ((seed_offset << 32) >> 32);

    // start and end from the seed block_id table
    end = d_index_table[seed];
    start = 0;
    if (seed > 0){
        start = d_index_table[seed-1];
    }

    for (int id1 = start; id1 < end; id1 += 1) {
        if(thread_id == 0){ 
            ref_loc   = d_pos_table[id1];
            query_loc = q_start;
            total_score = 0; 
        }

        //////////////////////////////////////////////////////////////////

        thread_score = 0;
        if(thread_id < seed_size){
            thread_score = sub_mat[d_ref_seq[ref_loc+thread_id]*5+d_query_seq[query_loc+thread_id]];
        }

        for (int offset = warp_size/2; offset > 0; offset /= 2)
            thread_score += __shfl_down_sync(0x13, thread_score, offset);

        if(thread_id == 0){
            total_score += thread_score;
        }
        __syncthreads();

        ////////////////////////////////////////////////////////////////

        tile = 0;
        right_xdrop_found = false;
        right_edge = false;
        prev_score = 0;
        prev_max_score = 0;

        while(tile < NUM_TILES && !right_xdrop_found && !right_edge){
            ref_pos   = ref_loc + seed_size + thread_id + tile*warp_size;
            query_pos = query_loc + seed_size + thread_id + tile*warp_size;

            if(ref_pos < ref_len && query_pos < query_len){
                thread_score = sub_mat[d_ref_seq[ref_pos]*5+d_query_seq[query_pos]];
            }

        #pragma unroll
            for (int offset = 1; offset < warp_size; offset *= 2){
                temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                if(thread_id >= offset){
                    thread_score += temp;
                }
            }

            thread_score += prev_score;
            max_thread_score = max(thread_score, prev_max_score);
            __syncthreads();

        #pragma unroll
            for (int offset = 1; offset < warp_size; offset *= 2){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);

                if(thread_id >= offset){
                    max_thread_score = max(max_thread_score, temp);
                }
            }

            xdrop_done = ((max_thread_score-thread_score) > xdrop);
            __syncthreads();

        #pragma unroll
            for (int offset = 1; offset < warp_size; offset *= 2){
                xdrop_done |= __shfl_up_sync(0xFFFFFFFF, xdrop_done, offset);
            }

            if(thread_id == warp_size-1){
                if(xdrop_done){
                    total_score+=max_thread_score;
                    right_xdrop_found = true;
                    right_extent = tile*BLOCK_SIZE-1;
                }
                else if(ref_pos > ref_len || query_pos > query_len)
                    right_edge = true;
                else{
                    prev_score = thread_score;
                    prev_max_score = max_thread_score;
                }
            }
            __syncthreads();
            
            tile++;
        }

        ////////////////////////////////////////////////////////////////

        tile = 0;
        left_xdrop_found = false;
        left_edge = false;
        prev_score = 0;
        prev_max_score = 0;

        while(tile < NUM_TILES && !left_xdrop_found && !left_edge){

            ref_pos   = ref_loc - thread_id - 1 - tile*warp_size;
            query_pos = query_loc - thread_id - 1 - tile*warp_size;

            if(ref_pos >= 0  && query_pos >= 0){
                thread_score = sub_mat[d_ref_seq[ref_pos]*5+d_query_seq[query_pos]];
            }

        #pragma unroll
            for (int offset = 1; offset < warp_size; offset *= 2){
                temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                if(thread_id >= offset){
                    thread_score += temp;
                }
            }

            thread_score += prev_score;
            max_thread_score = max(thread_score, prev_max_score);
            __syncthreads();

        #pragma unroll
            for (int offset = 1; offset < warp_size; offset *= 2){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);

                if(thread_id >= offset){
                    max_thread_score = max(max_thread_score, temp);
                }
            }

            xdrop_done = ((max_thread_score-thread_score) > xdrop);
            __syncthreads();

        #pragma unroll
            for (int offset = 1; offset < warp_size; offset *= 2){
                xdrop_done |= __shfl_up_sync(0xFFFFFFFF, xdrop_done, offset);
            }

            if(thread_id == warp_size-1){
                if(xdrop_done){
                    total_score+=max_thread_score;
                    left_xdrop_found = true;
                    left_extent = tile*BLOCK_SIZE-1;
                }
                else if(ref_pos < 0 || query_pos < 0)
                    left_edge = true;
                else{
                    prev_score = thread_score;
                    prev_max_score = max_thread_score;
                }
            }
            __syncthreads();

            tile++;
        }

        //////////////////////////////////////////////////////////////////


        if(thread_id == 0){
            int dram_address = seed_hit_num[block_id]-id1+start-1;

            if(!left_edge && !right_edge){
                if(right_xdrop_found && left_xdrop_found){
                    if(total_score >= xdrop_threshold){
                        d_r_starts[dram_address] = ref_loc - left_extent;
                        d_q_starts[dram_address] = query_loc - left_extent;
                        d_len[dram_address] = left_extent+right_extent+seed_size;
                        d_done[dram_address] = true;
                    }
                    else{
                        d_r_starts[dram_address] = 0;
                        d_q_starts[dram_address] = 0;
                        d_len[dram_address] = 1;
                        d_done[dram_address] = false;
                    }
                }
                else{
                    d_r_starts[dram_address] = ref_loc;
                    d_q_starts[dram_address] = query_loc;
                    d_len[dram_address] = seed_size;
                    d_done[dram_address] = false;
                }
            }
            else{
                d_r_starts[dram_address] = 0;
                d_q_starts[dram_address] = 0;
                d_len[dram_address] = 2;
                d_done[dram_address] = false;
            }
        }
        __syncthreads();
    }
}

int SeedAndFilter (std::vector<uint64_t> seed_offset_vector, bool rev){

    gpu_lock.lock();
    int ret = 0;
    hipError_t err;
    seed_size = 19;

    uint32_t num_hits;
    int total_anchors = 0;
    int total1 = 0;
    int total2 = 0;
    int total_seed = 0;

    uint32_t num_seeds = seed_offset_vector.size();
    assert(num_seeds <= 13*cfg.chunk_size);

    if (num_seeds == 0) {
        return ret;
    }

    for (uint32_t i = 0; i < num_seeds; i++) {
        h_seed_offsets[i] = seed_offset_vector[i];
    }

    err = hipMemcpy(d_seed_offsets, h_seed_offsets, num_seeds*sizeof(uint64_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }

    thrust::device_vector<int> seed_hit_num(num_seeds);
    int* seed_hit_num_array = thrust::raw_pointer_cast(&seed_hit_num[0]);

    find_num_hits <<<MAX_BLOCKS, MAX_THREADS>>> (num_seeds, d_index_table, d_seed_offsets, d_num_seed_hits, seed_hit_num_array);

    thrust::inclusive_scan(seed_hit_num.begin(), seed_hit_num.end(), seed_hit_num.begin());

    err = hipMemcpy(&num_hits, (seed_hit_num_array+num_seeds-1), sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed! SF1\n");
        exit(1);
    }

    uint32_t* h_r_starts = (uint32_t*) calloc(num_hits, sizeof(uint32_t));
    uint32_t* h_q_starts = (uint32_t*) calloc(num_hits, sizeof(uint32_t));
    uint32_t* h_len      = (uint32_t*) calloc(num_hits, sizeof(uint32_t));
    bool* h_done             = (bool*) calloc(num_hits, sizeof(bool));

    err = hipMalloc(&d_r_starts, num_hits*sizeof(uint32_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "2 Error: hipMalloc failed! SF2\n");
        exit(1);
    }

    err = hipMalloc(&d_q_starts, num_hits*sizeof(uint32_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "3 Error: hipMalloc failed! SF3\n");
        exit(1);
    }

    err = hipMalloc(&d_len, num_hits*sizeof(uint32_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "4 Error: hipMalloc failed! SF4\n");
        exit(1);
    }

    err = hipMalloc(&d_done, num_hits*sizeof(bool)); 
    if (err != hipSuccess) {
        fprintf(stderr, "5 Error: hipMalloc failed! SF5\n");
        exit(1);
    }

    gettimeofday(&time2, NULL);
//    printf("Start find_anchors %d\n", num_seeds);
    find_anchors3 <<<num_seeds,BLOCK_SIZE>>> (num_seeds, d_ref_seq, d_query_seq, d_index_table, d_pos_table, d_seed_offsets, d_sub_mat, cfg.xdrop, cfg.xdrop_threshold, d_r_starts, d_q_starts, d_len, d_done, ref_len, query_len, seed_size, seed_hit_num_array, num_hits);

    err = hipMemcpy(h_r_starts, d_r_starts, num_hits*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed! SF6\n");
        exit(1);
    }
    
    err = hipMemcpy(h_q_starts, d_q_starts, num_hits*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed! SF7\n");
        exit(1);
    }
    
    err = hipMemcpy(h_len, d_len, num_hits*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed! SF8\n");
        exit(1);
    }

    err = hipMemcpy(h_done, d_done, num_hits*sizeof(bool), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed! SF9\n");
        exit(1);
    }

    gettimeofday(&time4, NULL);

    for(int i = 0; i < num_hits; i++){
        if(h_done[i]){
            total_anchors++;
        }
        else{
            if(h_len[i] == 1)
                total1++;
            else if(h_len[i] == 2)
                total2++;
            else if(h_len[i] == seed_size)
                total_seed++;
        }
    }

    useconds1 = time4.tv_usec - time2.tv_usec;
    seconds1  = time4.tv_sec  - time2.tv_sec;
    mseconds1 = ((seconds1) * 1000 + useconds1/1000.0) + 0.5;
    fprintf(stdout, "%d %d %d %d %d %lu msec\n", total1, total2, total_seed, total_anchors, num_hits, mseconds1);

    free(h_r_starts);
    free(h_q_starts);
    free(h_len);
    free(h_done);
    hipFree(d_r_starts);
    hipFree(d_q_starts);
    hipFree(d_len);
    hipFree(d_done);

    gpu_lock.unlock();

    return ret;
}

size_t InitializeProcessor (int t, int f){
    size_t ret = 0;
    hipError_t err;

    err = hipMalloc(&d_seed_offsets, 13*cfg.chunk_size*sizeof(uint64_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "1 Error: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_num_seed_hits, 13*cfg.chunk_size*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "2 Error: hipMalloc failed!\n");
        exit(1);
    }


    sub_mat = (int *)malloc(25 * sizeof(int)); 

    sub_mat[24] = cfg.gact_sub_mat[10];
    for(int i = 0; i < 4; i++){
        sub_mat[i*5+4] = cfg.gact_sub_mat[10];
        sub_mat[4*5+i] = cfg.gact_sub_mat[10];
        sub_mat[i*5+i] = cfg.gact_sub_mat[i*4 + i - mat_offset[i]];
    }

    for(int i = 0; i < 4; i++){
        for(int j = i+1; j < 4; j++){
            sub_mat[i*5+j] = cfg.gact_sub_mat[i*4 + j - mat_offset[i]];
            sub_mat[j*5+i] = sub_mat[i*5+j];
        }
    }

    err = hipMalloc(&d_sub_mat, 25*sizeof(int)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_sub_mat, sub_mat, 25*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "1. Error: hipMemcpy failed!\n");
        exit(1);
    }

    return ret;
}

void SendRefWriteRequest (size_t start_addr, size_t len){
    hipError_t err;
    ref_len = len;
    
    char* d_ref_seq_tmp;
    err = hipMalloc(&d_ref_seq_tmp, len*sizeof(char)); 
    if (err != hipSuccess) {
        fprintf(stderr, "1 Error: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_ref_seq_tmp, g_DRAM->buffer + start_addr, len*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "1. Error: hipMemcpy failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_ref_seq, len*sizeof(char)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }
    
    compress_string <<<MAX_BLOCKS, MAX_THREADS>>> (len, d_ref_seq_tmp, d_ref_seq);
    
    hipFree(d_ref_seq_tmp);
}

void SendQueryWriteRequest (size_t start_addr, size_t len){
    hipError_t err;
    query_len = len;
    
    char* d_query_seq_tmp;
    err = hipMalloc(&d_query_seq_tmp, len*sizeof(char)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }
    err = hipMemcpy(d_query_seq_tmp, g_DRAM->buffer + start_addr, len*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }
    
    err = hipMalloc(&d_query_seq, len*sizeof(char)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }

    compress_string <<<MAX_BLOCKS, MAX_THREADS>>> (len, d_query_seq_tmp, d_query_seq);

    hipFree(d_query_seq_tmp);
}

void SendSeedPosTable (uint32_t* index_table, uint32_t index_table_size, uint64_t* pos_table, uint32_t num_index){
    hipError_t err;

    h_seed_offsets = (uint64_t*) malloc(13*cfg.chunk_size*sizeof(uint64_t));

    err = hipMalloc(&d_index_table, index_table_size*sizeof(uint32_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }
    err = hipMemcpy(d_index_table, index_table, index_table_size*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }
    err = hipMalloc(&d_pos_table, num_index*sizeof(uint64_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }
    err = hipMemcpy(d_pos_table, pos_table, num_index*sizeof(uint64_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }

    fprintf(stdout, "Sending seed position table to GPU successful.\n");
}

std::vector<tile_output> SendBatchRequest (std::vector<filter_tile> tiles, uint8_t align_fields, int thresh) {
        std::vector<tile_output> filtered_op;

        return filtered_op;
}

extend_output GACTXRequest (extend_tile tile, uint8_t align_fields) {
        extend_output op;

        return op;
}

void SendRequest (size_t ref_offset, size_t query_offset, size_t ref_length, size_t query_length, uint8_t align_fields){

}

void ShutdownProcessor(){
    hipFree(d_ref_seq);
    hipFree(d_query_seq);
    hipFree(d_seed_offsets);
    hipFree(d_index_table);
    hipFree(d_pos_table);
}

DRAM *g_DRAM = nullptr;

InitializeProcessor_ptr g_InitializeProcessor = InitializeProcessor;
ShutdownProcessor_ptr g_ShutdownProcessor = ShutdownProcessor;
SendRequest_ptr g_SendRequest = SendRequest;
SendSeedPosTable_ptr g_SendSeedPosTable = SendSeedPosTable;
SeedAndFilter_ptr g_SeedAndFilter = SeedAndFilter;
SendRefWriteRequest_ptr g_SendRefWriteRequest = SendRefWriteRequest;
SendQueryWriteRequest_ptr g_SendQueryWriteRequest = SendQueryWriteRequest;       
SendBatchRequest_ptr g_SendBatchRequest = SendBatchRequest;
GACTXRequest_ptr g_GACTXRequest = GACTXRequest;
