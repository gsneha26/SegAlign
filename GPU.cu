#include "hip/hip_runtime.h"
#include <thrust/scan.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "graph.h"
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <algorithm>
#include <mutex>
#include <cstring>
#include <cstdlib>
#include "tbb/scalable_allocator.h"
#include <stdio.h>
#include <stdlib.h>

#define MAX_BLOCKS 1<<16
#define MAX_THREADS 1024 
#define NUM_BLOCKS 2048
#define NUM_THREADS 64 
#define NUM_WARPS 2
#define HIT_LIMIT 512
#define OUT_LIMIT 20

std::mutex gpu_lock;

const char A_NT = 0;
const char C_NT = 1;
const char G_NT = 2;
const char T_NT = 3;
const char N_NT = 4;

int mat_offset[] = {0, 1, 3, 6};                                                             
int *d_sub_mat;

int err;                            
int check_status = 0;

int ref_len;
int query_len;
int seed_size;

struct timeval start_time1, end_time1; 
long useconds1, seconds1, mseconds1;

char* d_ref_seq;
char* d_query_seq;
uint64_t* d_seed_offsets;
uint32_t* d_index_table;
uint32_t* d_num_seed_hits;
uint64_t* d_pos_table;
uint32_t* d_r_starts;
uint32_t* d_q_starts;
uint32_t* d_len;
bool* d_done;
uint64_t* h_seed_offsets;
int *sub_mat;
int size_out;

__global__
void compress_string (uint32_t n, char* src_seq, char* dst_seq){ 
    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    uint32_t len = n;

    for (uint32_t i = start; i < len; i += stride) {
        char ch = src_seq[i];
        char dst = N_NT;
        if ((ch == 'a') || (ch == 'A'))
            dst = A_NT;
        else if ((ch == 'c') || (ch == 'C'))
            dst = C_NT;
        else if ((ch == 'g') || (ch == 'G'))
            dst = G_NT;
        else if ((ch == 't') || (ch == 'T'))
            dst = T_NT;
        dst_seq[i] = dst;
    }
}

__global__
void find_num_hits (int num_seeds, uint32_t* d_index_table, uint64_t* seed_offsets, uint32_t* d_num_seed_hits, int* seed_hit_num){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    uint32_t num_seed_hit;
    uint32_t seed;
    
    for (uint32_t id = start; id < num_seeds; id += stride) {
        seed = (seed_offsets[id] >> 32);

        // start and end from the seed block_id table
        num_seed_hit = d_index_table[seed];
        if (seed > 0){
            num_seed_hit -= d_index_table[seed-1];
        }

        d_num_seed_hits[id] = num_seed_hit;
        seed_hit_num[id] = num_seed_hit;
    }

}

__global__
void find_anchors (int num_seeds, char* d_ref_seq, char* d_query_seq, uint32_t* d_index_table, uint64_t* d_pos_table, uint64_t *seed_offsets, int *d_sub_mat, int xdrop, int xdrop_threshold, uint32_t* d_r_starts, uint32_t* d_q_starts, uint32_t* d_len, bool* d_done, int ref_len, int query_len, int seed_size, int* seed_hit_num, int num_hits){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    uint32_t start, end;
    uint32_t q_start;
    uint64_t seed_offset;
    uint32_t seed;

    __shared__ int ref_loc;
    __shared__ int query_loc;
    __shared__ int total_score;
    __shared__ int warp_size;
    int max_score;
    __shared__ bool right_edge; 
    __shared__ bool  left_edge; 
    __shared__ bool right_xdrop_found; 
    __shared__ bool left_xdrop_found; 
    __shared__ uint32_t left_extent;
    __shared__ uint32_t right_extent;
    __shared__ uint32_t hit_num[32];
    int thread_score;
    int current_id;
    int temp;

    __shared__ uint32_t r_starts[HIT_LIMIT];
    __shared__ uint32_t q_starts[HIT_LIMIT];
    __shared__ int score[NUM_THREADS];
    __shared__ int block_score[NUM_WARPS];
    __shared__ uint32_t final_hits;
    __shared__ uint32_t total_hits;
    __shared__ uint32_t total_curr_hits;
    __shared__ int sub_mat[25];
    __shared__ uint32_t total_anchors;
    __shared__ uint32_t total_not;

    if(thread_id < 25){
        sub_mat[thread_id] = d_sub_mat[thread_id];
    }

    if(thread_id == 0){
        total_hits = 0;
        final_hits = 0;
        total_anchors = 0;
        total_not = 0;
        total_curr_hits = 0;
        warp_size = warpSize;
    }
    __syncthreads();

    int id_start = warp_size * block_id;
    int stride = grid_dim * warp_size;
    int lane_id = thread_id % warp_size;
    int warp_num = (thread_id - lane_id)/warp_size;
    __syncthreads();

    for (int id = id_start; id < num_seeds; id = id+ stride) {

        if(thread_id < warp_size){
            current_id = id + thread_id;

            if(current_id < num_seeds){
                seed_offset = seed_offsets[current_id];

                seed = (seed_offset >> 32);
                q_start = ((seed_offset << 32) >> 32);

                // start and end from the seed block_id table
                end = d_index_table[seed];
                start = 0;
                if (seed > 0){
                    start = d_index_table[seed-1];
                }
            }
            else{
                start = 0;
                end = 0;
            }

            hit_num[thread_id] = end-start;
        }
        __syncthreads();

//        int k_val;
//        for(int k = 1; k < block_dim; k=k*2){
//            if(thread_id >= k){
//                k_val = hit_num[thread_id-k];
//            }
//            __syncthreads();
//
//            if(thread_id >= k){
//                hit_num[thread_id] += k_val;
//            }
//            __syncthreads();
//        }

        if(thread_id == 0){
            for(int i = 1; i < warp_size; i++)
                hit_num[i] += hit_num[i-1];

            total_hits = hit_num[warp_size-1];
            final_hits += total_hits; 
        }
        __syncthreads();

        for(int hit_limit = 0; hit_limit < total_hits; hit_limit=hit_limit+HIT_LIMIT){
            if(thread_id < warp_size){
                int addr_start = (thread_id == 0) ? 0 : hit_num[thread_id-1];
                for (uint32_t p = start; p < end; p++) { 
                    int index_el = addr_start+p-start;
                    if ((index_el >= hit_limit) && (index_el < (hit_limit+HIT_LIMIT))) { 
                        r_starts[index_el-hit_limit] = d_pos_table[p];
                        q_starts[index_el-hit_limit] = q_start;
                    }
                }
            }
            __syncthreads();

            if(thread_id == 0){
                if(total_hits > hit_limit+HIT_LIMIT){
                    total_curr_hits = HIT_LIMIT;
                }
                else{
                    total_curr_hits = total_hits-hit_limit;
                }
            }
            __syncthreads();

            for (int id1 = 0; id1 < total_curr_hits; id1 += 1) {
                ref_loc   = r_starts[id1];
                query_loc = q_starts[id1];
                total_score = 0;

                //////////////////////////////////////////////////////////////////
                
                thread_score = 0;
                if(thread_id < seed_size){
                    thread_score = sub_mat[d_ref_seq[ref_loc+thread_id]*5+d_query_seq[query_loc+thread_id]];
                }

                for (int offset = warp_size/2; offset > 0; offset /= 2)
                    thread_score += __shfl_down_sync(0x13, thread_score, offset);

                if(thread_id == 0){
                    total_score += thread_score;
                }

                //////////////////////////////////////////////////////////////////

                thread_score = 0;
                if(ref_loc+seed_size+thread_id < ref_len && query_loc+seed_size+thread_id < query_len){
                    thread_score = sub_mat[d_ref_seq[ref_loc+seed_size+thread_id]*5+d_query_seq[query_loc+seed_size+thread_id]];
                }

                for (int offset = 1; offset < warp_size; offset *= 2){
                    temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                    if(lane_id >= offset){
                        thread_score += temp;
                    }
                }
                __syncthreads();

//                if(lane_id == warp_size-1){
//                    block_score[warp_num] = thread_score;
//                }
//                __syncthreads();
//
//                int block_sum;
//                if(thread_id < NUM_WARPS){
//                    block_sum = block_score[thread_id]; 
//                    for (int offset = 1; offset < NUM_WARPS; offset *= 2){
//                        temp = __shfl_up_sync(0xFFFFFFFF, block_sum, offset);
//
//                        if(lane_id >= offset){
//                            block_sum += temp;
//                        }
//                    }
//                    block_score[thread_id] = block_sum;
//                }
//                __syncthreads();
//
//                if(warp_num > 0){
//                    thread_score +=  block_score[warp_num-1];
//                }
//                __syncthreads();

                score[thread_id] = thread_score;
                __syncthreads();

                for(int i = NUM_WARPS-1; i > 0; i--){
                    if(thread_id >= warp_size*i){
                        score[thread_id] += score[i*warp_size-1];
                    }
                    __syncthreads();
                }

                if(thread_id == 0){
                    right_xdrop_found = false;
                    right_edge = false;
                    max_score = 0;
                    for(int i = 0; i < NUM_THREADS; i++){
                        if(score[i] > max_score){
                           max_score = score[i];
                        }

                        if(max_score-score[i] > xdrop && right_xdrop_found == false){
                            total_score+=max_score;
                            right_xdrop_found = true;
                            right_extent = i;
                            break;
                        }
                    }

                    if(right_xdrop_found == false && (ref_loc + NUM_THREADS > ref_len || query_loc + NUM_THREADS > query_len))
                        right_edge = true;
                }
                __syncthreads();

                //////////////////////////////////////////////////////////////////

                thread_score = 0;
                if(ref_loc-thread_id-1 >= 0 && query_loc-thread_id-1 >=0){
                    thread_score = sub_mat[d_ref_seq[ref_loc-thread_id-1]*5+d_query_seq[query_loc-thread_id-1]];
                }

                for (int offset = 1; offset < warp_size; offset *= 2){
                    temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                    if(lane_id >= offset){
                        thread_score += temp;
                    }
                }

                score[thread_id] = thread_score;
                __syncthreads();

                for(int i = NUM_WARPS-1; i > 0; i--){
                    if(thread_id >= warp_size*i){
                        score[thread_id] += score[i*warp_size-1];
                    }
                    __syncthreads();
                }

                if(thread_id == 0){
                    left_xdrop_found = false;
                    left_edge = false;
                    max_score = 0;
                    for(int i = 0; i < NUM_THREADS; i++){
                        if(score[i] > max_score){
                           max_score = score[i];
                        }

                        if(max_score-score[i] > xdrop && left_xdrop_found == false){
                            total_score+=max_score;
                            left_xdrop_found = true;
                            left_extent = i;
                            break;
                        }
                    }

                    if(left_xdrop_found == false && (ref_loc < NUM_THREADS || query_loc < NUM_THREADS))
                        left_edge = true;
                }
                __syncthreads();

                //////////////////////////////////////////////////////////////////

                if(thread_id == 0){
                    if(!left_edge && !right_edge){
//                    if(total_anchors < OUT_LIMIT && !left_edge && !right_edge){
                        if(right_xdrop_found && left_xdrop_found){
                            if(total_score >= xdrop_threshold){
//                                d_r_starts[block_id*OUT_LIMIT + total_anchors] = ref_loc - left_extent;
//                                d_q_starts[block_id*OUT_LIMIT + total_anchors] = query_loc - left_extent;
//                                d_len[block_id*OUT_LIMIT + total_anchors] = left_extent+right_extent+seed_size;
//                                d_done[block_id*OUT_LIMIT + total_anchors] = true;
                                total_anchors++;
                            }
                        }
                        else{
//                            d_r_starts[block_id*OUT_LIMIT + total_anchors] = ref_loc;
//                            d_q_starts[block_id*OUT_LIMIT + total_anchors] = query_loc;
//                            d_len[block_id*OUT_LIMIT + total_anchors] = seed_size;
//                            d_done[block_id*OUT_LIMIT + total_anchors] = false;
//                            total_anchors++;
                            total_not++;
                        }
                    }
                }
            __syncthreads();
            }
        }
    }
    d_r_starts[block_id] = total_anchors;
    d_q_starts[block_id] =  total_not;
    d_len[block_id] = final_hits;
}

__global__
void find_anchors1 (int num_seeds, char* d_ref_seq, char* d_query_seq, uint32_t* d_index_table, uint64_t* d_pos_table, uint64_t* d_seed_offsets, int *d_sub_mat, int xdrop, int xdrop_threshold, uint32_t* d_r_starts, uint32_t* d_q_starts, uint32_t* d_len, bool* d_done, int ref_len, int query_len, int seed_size, int* seed_hit_num, int num_hits){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;

    __shared__ uint32_t start, end;
    __shared__ uint32_t q_start;
    __shared__ uint32_t seed;
    __shared__ uint64_t seed_offset;

    __shared__ uint32_t ref_loc;
    __shared__ uint32_t query_loc;
    __shared__ int total_score;
    __shared__ int total_score1;
    __shared__ int warp_size;
    int max_score;
    __shared__ bool right_edge; 
    __shared__ bool  left_edge; 
    __shared__ bool right_xdrop_found; 
    __shared__ bool  left_xdrop_found; 
    __shared__ uint32_t left_extent;
    __shared__ uint32_t right_extent;
    int thread_score;
    int temp;

    __shared__ int score[NUM_THREADS];
    __shared__ int sub_mat[25];

    if(thread_id < 25){
        sub_mat[thread_id] = d_sub_mat[thread_id];
    }

    if(thread_id == 0){
        warp_size = warpSize;
        seed_offset = d_seed_offsets[block_id];
    }

    int lane_id = thread_id % warp_size;
    __syncthreads();

    seed = (seed_offset >> 32);
    q_start = ((seed_offset << 32) >> 32);

    // start and end from the seed block_id table
    end = d_index_table[seed];
    start = 0;
    if (seed > 0){
        start = d_index_table[seed-1];
    }
    else{
        start = 0;
        end = 0;
    }

    for (int id1 = start; id1 < end; id1 += 1) {
        ref_loc   = d_pos_table[id1];
        query_loc = q_start;
        total_score = 0; 

        //////////////////////////////////////////////////////////////////

        thread_score = 0;
        if(thread_id < seed_size){
            thread_score = sub_mat[d_ref_seq[ref_loc+thread_id]*5+d_query_seq[query_loc+thread_id]];
        }

        for (int offset = warp_size/2; offset > 0; offset /= 2)
            thread_score += __shfl_down_sync(0x13, thread_score, offset);

        if(thread_id == 0){
            total_score += thread_score;
            total_score1 = thread_score;
        }
        __syncthreads();

        //////////////////////////////////////////////////////////////////

        thread_score = 0;
        if(ref_loc+seed_size+thread_id < ref_len && query_loc+seed_size+thread_id < query_len){
            thread_score = sub_mat[d_ref_seq[ref_loc+seed_size+thread_id]*5+d_query_seq[query_loc+seed_size+thread_id]];
        }

        for (int offset = 1; offset < warp_size; offset *= 2){
            temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

            if(lane_id >= offset){
                thread_score += temp;
            }
        }
        __syncthreads();

        score[thread_id] = thread_score;
        __syncthreads();

        for(int i = NUM_WARPS-1; i > 0; i--){
            if(thread_id >= warp_size*i){
                score[thread_id] += score[i*warp_size-1];
            }
            __syncthreads();
        }

        if(thread_id == 0){
            right_xdrop_found = false;
            right_edge = false;
            max_score = 0;
            for(int i = 0; i < NUM_THREADS; i++){
                if(score[i] > max_score){
                    max_score = score[i];
                }

                if(max_score-score[i] > xdrop && right_xdrop_found == false){
                    total_score+=max_score;
                    right_xdrop_found = true;
                    right_extent = i;
                    break;
                }
            }

            if(right_xdrop_found == false && (ref_loc + NUM_THREADS > ref_len || query_loc + NUM_THREADS > query_len))
                right_edge = true;
        }
        __syncthreads();

        //////////////////////////////////////////////////////////////////

        thread_score = 0;
        if(ref_loc >= thread_id+1 && query_loc>= thread_id+1){
            thread_score = sub_mat[d_ref_seq[ref_loc-thread_id-1]*5+d_query_seq[query_loc-thread_id-1]];
        }

        for (int offset = 1; offset < warp_size; offset *= 2){
            temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

            if(lane_id >= offset){
                thread_score += temp;
            }
        }

        score[thread_id] = thread_score;
        __syncthreads();

        for(int i = NUM_WARPS-1; i > 0; i--){
            if(thread_id >= warp_size*i){
                score[thread_id] += score[i*warp_size-1];
            }
            __syncthreads();
        }

        if(thread_id == 0){
            left_xdrop_found = false;
            left_edge = false;
            max_score = 0;
            for(int i = 0; i < NUM_THREADS; i++){
                if(score[i] > max_score){
                    max_score = score[i];
                }

                if(max_score-score[i] > xdrop && left_xdrop_found == false){
                    total_score+=max_score;
                    left_xdrop_found = true;
                    left_extent = i;
                    break;
                }
            }

            if(left_xdrop_found == false && (ref_loc < NUM_THREADS || query_loc < NUM_THREADS))
                left_edge = true;
        }
        __syncthreads();

        //////////////////////////////////////////////////////////////////

        if(thread_id == 0){
            if(!left_edge && !right_edge){
                if(right_xdrop_found && left_xdrop_found){
                    if(total_score >= xdrop_threshold){
                        d_r_starts[seed_hit_num[block_id]-id1+start-1] = ref_loc - left_extent;
                        d_q_starts[seed_hit_num[block_id]-id1+start-1] = query_loc - left_extent;
                        d_len[seed_hit_num[block_id]-id1+start-1] = left_extent+right_extent+seed_size;
                        d_done[seed_hit_num[block_id]-id1+start-1] = true;
                    }
                    else{
                        d_r_starts[seed_hit_num[block_id]-id1+start-1] = 0;
                        d_q_starts[seed_hit_num[block_id]-id1+start-1] = 0;
                        d_len[seed_hit_num[block_id]-id1+start-1] = 1;
                        d_done[seed_hit_num[block_id]-id1+start-1] = false;
                    }
                }
                else{
                    d_r_starts[seed_hit_num[block_id]-id1+start-1] = ref_loc;
                    d_q_starts[seed_hit_num[block_id]-id1+start-1] = query_loc;
                    d_len[seed_hit_num[block_id]-id1+start-1] = seed_size;
                    d_done[seed_hit_num[block_id]-id1+start-1] = false;
                }
            }
            else{
                d_r_starts[seed_hit_num[block_id]-id1+start-1] = 0;
                d_q_starts[seed_hit_num[block_id]-id1+start-1] = 0;
                d_len[seed_hit_num[block_id]-id1+start-1] = 2;
                d_done[seed_hit_num[block_id]-id1+start-1] = false;
            }
        }
        __syncthreads();
    }
}

__global__
void find_anchors2 (int num_seeds, char* d_ref_seq, char* d_query_seq, uint32_t* d_index_table, uint64_t* d_pos_table, uint64_t* d_seed_offsets, int *d_sub_mat, int xdrop, int xdrop_threshold, uint32_t* d_r_starts, uint32_t* d_q_starts, uint32_t* d_len, bool* d_done, int ref_len, int query_len, int seed_size, int* seed_hit_num, int num_hits){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;

    __shared__ uint32_t start, end;
    __shared__ uint32_t q_start;
    __shared__ uint64_t seed_offset;
    __shared__ uint32_t seed;

    __shared__ uint32_t ref_loc;
    __shared__ uint32_t query_loc;
    __shared__ int total_score;
    __shared__ int warp_size;
    int max_score;
    __shared__ bool right_edge; 
    __shared__ bool  left_edge; 
    __shared__ bool right_xdrop_found; 
    __shared__ bool  left_xdrop_found; 
    __shared__ uint32_t left_extent;
    __shared__ uint32_t right_extent;
    int thread_score;

    __shared__ int score[NUM_THREADS];
    __shared__ int sub_mat[25];
    __shared__ uint32_t total_anchors;

    if(thread_id < 25){
        sub_mat[thread_id] = d_sub_mat[thread_id];
    }

    if(thread_id == 0){
        total_anchors = 0;
        warp_size = warpSize;
        seed_offset = d_seed_offsets[block_id];
    }

    int lane_id = thread_id % warp_size;
    __syncthreads();

    seed = (seed_offset >> 32);
    q_start = ((seed_offset << 32) >> 32);

    // start and end from the seed block_id table
    end = d_index_table[seed];
    start = 0;
    if (seed > 0){
        start = d_index_table[seed-1];
    }
    else{
        start = 0;
        end = 0;
    }

    for (int id1 = start; id1 < end; id1 += 1) {
        ref_loc   = d_pos_table[id1];
        query_loc = q_start;
        total_score = 0; 

        //////////////////////////////////////////////////////////////////

        thread_score = 0;
        if(thread_id < seed_size){
            thread_score = sub_mat[d_ref_seq[ref_loc+thread_id]*5+d_query_seq[query_loc+thread_id]];
        }

        for (int offset = warp_size/2; offset > 0; offset /= 2)
            thread_score += __shfl_down_sync(0x13, thread_score, offset);

        if(thread_id == 0){
            total_score += thread_score;
        }

        //////////////////////////////////////////////////////////////////

        thread_score = 0;
        if(ref_loc+seed_size+thread_id < ref_len && query_loc+seed_size+thread_id < query_len){
            score[thread_id] = sub_mat[d_ref_seq[ref_loc+seed_size+thread_id]*5+d_query_seq[query_loc+seed_size+thread_id]];
        }
        __syncthreads();

        thrust::inclusive_scan(thrust::device, score, score+NUM_THREADS-1, score);
//        //parallel prefix sum
//        int k_val;
//        for(int k = 1; k < NUM_THREADS; k=k*2){
//            if(thread_id >= k){
//                k_val = score[thread_id-k];
//            }
//            __syncthreads();
//            if(thread_id >= k){
//                score[thread_id] += k_val;
//            }
//            __syncthreads();
//        }

        if(thread_id == 0){
            right_xdrop_found = false;
            right_edge = false;
            max_score = 0;
            for(int i = 0; i < NUM_THREADS; i++){
                if(score[i] > max_score){
                    max_score = score[i];
                }

                if(max_score-score[i] > xdrop && right_xdrop_found == false){
                    total_score+=max_score;
                    right_xdrop_found = true;
                    right_extent = i;
                    break;
                }
            }

            if(right_xdrop_found == false && (ref_loc + NUM_THREADS > ref_len || query_loc + NUM_THREADS > query_len))
                right_edge = true;
        }
        __syncthreads();

        //////////////////////////////////////////////////////////////////

        thread_score = 0;
        if(ref_loc >= thread_id+1 && query_loc>= thread_id+1){
            score[thread_id] = sub_mat[d_ref_seq[ref_loc-thread_id-1]*5+d_query_seq[query_loc-thread_id-1]];
        }
        __syncthreads();

        thrust::inclusive_scan(thrust::device, score, score+NUM_THREADS-1, score);
//        //parallel prefix sum
//        for(int k = 1; k < NUM_THREADS; k=k*2){
//            if(thread_id >= k){
//                k_val = score[thread_id-k];
//            }
//            __syncthreads();
//            if(thread_id >= k){
//                score[thread_id] += k_val;
//            }
//            __syncthreads();
//        }

        if(thread_id == 0){
            left_xdrop_found = false;
            left_edge = false;
            max_score = 0;
            for(int i = 0; i < NUM_THREADS; i++){
                if(score[i] > max_score){
                    max_score = score[i];
                }

                if(max_score-score[i] > xdrop && left_xdrop_found == false){
                    total_score+=max_score;
                    left_xdrop_found = true;
                    left_extent = i;
                    break;
                }
            }

            if(left_xdrop_found == false && (ref_loc < NUM_THREADS || query_loc < NUM_THREADS))
                left_edge = true;
        }
        __syncthreads();

        //////////////////////////////////////////////////////////////////

        if(thread_id == 0){
            if(!left_edge && !right_edge){
                if(right_xdrop_found && left_xdrop_found){
                    if(total_score >= xdrop_threshold){
                        d_r_starts[seed_hit_num[block_id]-id1+start-1] = ref_loc - left_extent;
                        d_q_starts[seed_hit_num[block_id]-id1+start-1] = query_loc - left_extent;
                        d_len[seed_hit_num[block_id]-id1+start-1] = left_extent+right_extent+seed_size;
                        d_done[seed_hit_num[block_id]-id1+start-1] = true;
                    }
                    else{
                        d_r_starts[seed_hit_num[block_id]-id1+start-1] = 0;
                        d_q_starts[seed_hit_num[block_id]-id1+start-1] = 0;
                        d_len[seed_hit_num[block_id]-id1+start-1] = 0;
                        d_done[seed_hit_num[block_id]-id1+start-1] = false;
                    }
                }
                else{
                    d_r_starts[seed_hit_num[block_id]-id1+start-1] = ref_loc;
                    d_q_starts[seed_hit_num[block_id]-id1+start-1] = query_loc;
                    d_len[seed_hit_num[block_id]-id1+start-1] = seed_size;
                    d_done[seed_hit_num[block_id]-id1+start-1] = false;
                }
            }
            else{
                d_r_starts[seed_hit_num[block_id]-id1+start-1] = 0;
                d_q_starts[seed_hit_num[block_id]-id1+start-1] = 0;
                d_len[seed_hit_num[block_id]-id1+start-1] = 0;
                d_done[seed_hit_num[block_id]-id1+start-1] = false;
            }
        }
        __syncthreads();
    }
}

int SeedAndFilter (std::vector<uint64_t> seed_offset_vector, bool rev){

    int ret = 0;
    hipError_t err;
    seed_size = 19;

    uint32_t num_seeds = seed_offset_vector.size();
    assert(num_seeds <= 13*cfg.chunk_size);
    if (num_seeds == 0) {
        return ret;
    }

    for (uint32_t i = 0; i < num_seeds; i++) {
        h_seed_offsets[i] = seed_offset_vector[i];
    }

    err = hipMemcpy(d_seed_offsets, h_seed_offsets, num_seeds*sizeof(uint64_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }

    thrust::device_vector<int> seed_hit_num(num_seeds);
    int* seed_hit_num_array = thrust::raw_pointer_cast(&seed_hit_num[0]);

    find_num_hits <<<MAX_BLOCKS, MAX_THREADS>>> (num_seeds, d_index_table, d_seed_offsets, d_num_seed_hits, seed_hit_num_array);

    thrust::inclusive_scan(seed_hit_num.begin(), seed_hit_num.end(), seed_hit_num.begin());

    uint32_t num_hits;
    err = hipMemcpy(&num_hits, (seed_hit_num_array+num_seeds-1), sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }

//    printf("num_seeds = %d %d\n", num_seeds, num_hits); 
    
    uint32_t* h_r_starts = (uint32_t*) calloc(num_hits, sizeof(uint32_t));
    uint32_t* h_q_starts = (uint32_t*) calloc(num_hits, sizeof(uint32_t));
    uint32_t* h_len      = (uint32_t*) calloc(num_hits, sizeof(uint32_t));
    bool* h_done             = (bool*) calloc(num_hits, sizeof(bool));

//    printf("Start find_anchors %d\n", num_seeds);
    find_anchors1 <<<num_seeds, NUM_THREADS>>> (num_seeds, d_ref_seq, d_query_seq, d_index_table, d_pos_table, d_seed_offsets, d_sub_mat, cfg.xdrop, cfg.xdrop_threshold, d_r_starts, d_q_starts, d_len, d_done, ref_len, query_len, seed_size, seed_hit_num_array, num_hits);
//    find_anchors2 <<<num_seeds, NUM_THREADS>>> (num_seeds, d_ref_seq, d_query_seq, d_index_table, d_pos_table, d_seed_offsets, d_sub_mat, cfg.xdrop, cfg.xdrop_threshold, d_r_starts, d_q_starts, d_len, d_done, ref_len, query_len, seed_size, seed_hit_num_array, num_hits);
//    find_anchors <<<NUM_BLOCKS, NUM_THREADS>>> (num_seeds, d_ref_seq, d_query_seq, d_index_table, d_pos_table, d_seed_offsets, d_sub_mat, cfg.xdrop, cfg.xdrop_threshold, d_r_starts, d_q_starts, d_len, d_done, ref_len, query_len, seed_size, seed_hit_num_array, num_hits);

    int num_hits1 = 2000;
    err = hipMemcpy(h_r_starts, d_r_starts, num_hits1*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }
    
    err = hipMemcpy(h_q_starts, d_q_starts, num_hits1*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }
    
    err = hipMemcpy(h_len, d_len, num_hits1*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(h_done, d_done, num_hits1*sizeof(bool), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }


    int total_anchors = 0;
    int total1 = 0;
    int total2 = 0;
    int total_seed = 0;

//    for(int i = 0; i < num_hits; i++){
//        printf("%d %d %d %d %d\n", num_seeds, i, h_r_starts[i], h_q_starts[i], h_len[i]);
//    }

    gettimeofday(&start_time1, NULL);

    for(int i = 0; i < num_hits; i++){
        if(h_done[i]){
            total_anchors++;
        }
        else{
            if(h_len[i] == 1)
                total1++;
            else if(h_len[i] == 2)
                total2++;
            else if(h_len[i] == seed_size)
                total_seed++;
        }
    }

    gettimeofday(&end_time1, NULL);
    useconds1 = end_time1.tv_usec - start_time1.tv_usec;
    seconds1  = end_time1.tv_sec  - start_time1.tv_sec;
    mseconds1 = ((seconds1) * 1000 + useconds1/1000.0) + 0.5;
    fprintf(stdout, "Time elapsed (loading query): %ld msec \n", mseconds1);

//    printf("%d %d %d %d %d\n", total1, total2, total_seed, total_anchors, num_hits);

    free(h_r_starts);
    free(h_q_starts);
    free(h_len);
    free(h_done);

    gpu_lock.unlock();

    return ret;
}

size_t InitializeProcessor (int t, int f){
    size_t ret = 0;
    hipError_t err;

    err = hipMalloc(&d_seed_offsets, 13*cfg.chunk_size*sizeof(uint64_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "1 Error: hipMalloc failed!\n");
        exit(1);
    }

//    size_out = NUM_BLOCKS*OUT_LIMIT;
    size_out = 13*cfg.chunk_size*OUT_LIMIT;

    err = hipMalloc(&d_num_seed_hits, 13*cfg.chunk_size*sizeof(uint32_t));
    if (err != hipSuccess) {
        fprintf(stderr, "2 Error: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_r_starts, size_out*sizeof(uint32_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "2 Error: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_q_starts, size_out*sizeof(uint32_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "3 Error: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_len, size_out*sizeof(uint32_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "4 Error: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_done, size_out*sizeof(bool)); 
    if (err != hipSuccess) {
        fprintf(stderr, "5 Error: hipMalloc failed!\n");
        exit(1);
    }

    sub_mat = (int *)malloc(25 * sizeof(int)); 

    sub_mat[24] = cfg.gact_sub_mat[10];
    for(int i = 0; i < 4; i++){
        sub_mat[i*5+4] = cfg.gact_sub_mat[10];
        sub_mat[4*5+i] = cfg.gact_sub_mat[10];
        sub_mat[i*5+i] = cfg.gact_sub_mat[i*4 + i - mat_offset[i]];
    }

    for(int i = 0; i < 4; i++){
        for(int j = i+1; j < 4; j++){
            sub_mat[i*5+j] = cfg.gact_sub_mat[i*4 + j - mat_offset[i]];
            sub_mat[j*5+i] = sub_mat[i*5+j];
        }
    }

    err = hipMalloc(&d_sub_mat, 25*sizeof(int)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_sub_mat, sub_mat, 25*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "1. Error: hipMemcpy failed!\n");
        exit(1);
    }

    return ret;
}

void SendRefWriteRequest (size_t start_addr, size_t len){
    hipError_t err;
    ref_len = len;
    
    char* d_ref_seq_tmp;
    err = hipMalloc(&d_ref_seq_tmp, len*sizeof(char)); 
    if (err != hipSuccess) {
        fprintf(stderr, "1 Error: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_ref_seq_tmp, g_DRAM->buffer + start_addr, len*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "1. Error: hipMemcpy failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_ref_seq, len*sizeof(char)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }
    
    compress_string <<<MAX_BLOCKS, MAX_THREADS>>> (len, d_ref_seq_tmp, d_ref_seq);
    
    hipFree(d_ref_seq_tmp);
}

void SendQueryWriteRequest (size_t start_addr, size_t len){
    hipError_t err;
    query_len = len;
    
    char* d_query_seq_tmp;
    err = hipMalloc(&d_query_seq_tmp, len*sizeof(char)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }
    err = hipMemcpy(d_query_seq_tmp, g_DRAM->buffer + start_addr, len*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }
    
    err = hipMalloc(&d_query_seq, len*sizeof(char)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }

    compress_string <<<MAX_BLOCKS, MAX_THREADS>>> (len, d_query_seq_tmp, d_query_seq);

    hipFree(d_query_seq_tmp);
}

void SendSeedPosTable (uint32_t* index_table, uint32_t index_table_size, uint64_t* pos_table, uint32_t num_index){
    hipError_t err;

    h_seed_offsets = (uint64_t*) malloc(13*cfg.chunk_size*sizeof(uint64_t));

    err = hipMalloc(&d_index_table, index_table_size*sizeof(uint32_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }
    err = hipMemcpy(d_index_table, index_table, index_table_size*sizeof(uint32_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }
    err = hipMalloc(&d_pos_table, num_index*sizeof(uint64_t)); 
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed!\n");
        exit(1);
    }
    err = hipMemcpy(d_pos_table, pos_table, num_index*sizeof(uint64_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed!\n");
        exit(1);
    }

    fprintf(stdout, "Sending seed position table to GPU successful.\n");
}

std::vector<tile_output> SendBatchRequest (std::vector<filter_tile> tiles, uint8_t align_fields, int thresh) {
        std::vector<tile_output> filtered_op;

        return filtered_op;
}

extend_output GACTXRequest (extend_tile tile, uint8_t align_fields) {
        extend_output op;

        return op;
}

void SendRequest (size_t ref_offset, size_t query_offset, size_t ref_length, size_t query_length, uint8_t align_fields){

}

void ShutdownProcessor(){
    hipFree(d_ref_seq);
    hipFree(d_query_seq);
    hipFree(d_seed_offsets);
    hipFree(d_index_table);
    hipFree(d_pos_table);
    hipFree(d_r_starts);
    hipFree(d_q_starts);
    hipFree(d_len);
    hipFree(d_done);
}

DRAM *g_DRAM = nullptr;

InitializeProcessor_ptr g_InitializeProcessor = InitializeProcessor;
ShutdownProcessor_ptr g_ShutdownProcessor = ShutdownProcessor;
SendRequest_ptr g_SendRequest = SendRequest;
SendSeedPosTable_ptr g_SendSeedPosTable = SendSeedPosTable;
SeedAndFilter_ptr g_SeedAndFilter = SeedAndFilter;
SendRefWriteRequest_ptr g_SendRefWriteRequest = SendRefWriteRequest;
SendQueryWriteRequest_ptr g_SendQueryWriteRequest = SendQueryWriteRequest;       
SendBatchRequest_ptr g_SendBatchRequest = SendBatchRequest;
GACTXRequest_ptr g_GACTXRequest = GACTXRequest;
