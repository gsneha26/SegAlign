#include "hip/hip_runtime.h"
#include "GPU.h"
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <iostream>
#include <thrust/scan.h>
#include <thrust/unique.h>
#include <thrust/find.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>

std::mutex gpu_lock;

int err;                            
int check_status = 0;
int NUM_DEVICES;
int MAX_SEEDS;
int MAX_HITS;
int MAX_HITS_SIZE;
std::vector<int> available_gpus;
std::mutex mu;
std::condition_variable cv;

uint32_t ref_len;
uint32_t query_length[BUFFER_DEPTH];

char** d_ref_seq;
char** d_query_seq;
char** d_query_rc_seq;

int **d_sub_mat;

uint32_t** d_index_table;
uint32_t** d_pos_table;

uint64_t** d_seed_offsets;

hsp** d_hsp;
hsp** d_hsp_reduced;
std::vector<thrust::device_vector<hsp> > d_hsp_vec;
std::vector<thrust::device_vector<hsp> > d_hsp_reduced_vec;

std::vector<thrust::device_vector<uint32_t> > d_done_vec;
std::vector<thrust::device_vector<uint32_t> > d_hit_num_vec;
uint32_t** d_done_array;
uint32_t** d_hit_num_array;

struct hspEqual{
    __host__ __device__
        bool operator()(hsp x, hsp y){
        return ((x.ref_start == y.ref_start) && (x.query_start == y.query_start) && (x.len == y.len) && (x.score == y.score));
    }
};

// wrap of hipMalloc error checking in one place.  
static inline void check_cuda_malloc(void** buf, size_t bytes, const char* tag) {
    hipError_t err = hipMalloc(buf, bytes);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc of %lu bytes for %s failed\n", bytes, tag);
        exit(1);
    }
}
	 
__global__
void compress_string_rev_comp (uint32_t len, char* src_seq, char* dst_seq, char* dst_seq_rc){ 
    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    for (uint32_t i = start; i < len; i += stride) {
        char ch = src_seq[i];
        char dst = X_NT;
        char dst_rc = X_NT;
        if (ch == 'A'){
            dst = A_NT;
            dst_rc = T_NT;
        }
        else if (ch == 'C'){ 
            dst = C_NT;
            dst_rc = G_NT;
        }
        else if (ch == 'G'){
            dst = G_NT;
            dst_rc = C_NT;
        }
        else if (ch == 'T'){
            dst = T_NT;
            dst_rc = A_NT;
        }
        else if ((ch == 'a') || (ch == 'c') || (ch == 'g') || (ch == 't')){
            dst = L_NT;
            dst_rc = L_NT;
        }
        else if ((ch == 'n') || (ch == 'N')){
            dst = N_NT;
            dst_rc = N_NT;
        }
        dst_seq[i] = dst;
        dst_seq_rc[len -1 -i] = dst_rc;
    }
}

__global__
void compress_string (uint32_t len, char* src_seq, char* dst_seq){ 
    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    for (uint32_t i = start; i < len; i += stride) {
        char ch = src_seq[i];
        char dst = X_NT;
        if (ch == 'A')
            dst = A_NT;
        else if (ch == 'C')
            dst = C_NT;
        else if (ch == 'G')
            dst = G_NT;
        else if (ch == 'T')
            dst = T_NT;
        else if ((ch == 'a') || (ch == 'c') || (ch == 'g') || (ch == 't'))
            dst = L_NT;
        else if ((ch == 'n') || (ch == 'N'))
            dst = N_NT;
        dst_seq[i] = dst;
    }
}

__global__
void fill_output (uint32_t* d_done, hsp* d_hsp, hsp* d_hsp_reduced, int num_hits){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;
    int index = 0;

    for (uint32_t id = start; id < num_hits; id += stride) {
        index = d_done[id];

        if(id > 0){
            if(index > d_done[id-1]){
                d_hsp_reduced[index-1]    =  d_hsp[id];
            }
        }
        else{
            if(index == 1){
                d_hsp_reduced[0]    = d_hsp[0];
            }
        }
    }
}

__global__
void find_num_hits (int num_seeds, const uint32_t* __restrict__ d_index_table, uint64_t* seed_offsets, uint32_t* seed_hit_num){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    uint32_t num_seed_hit;
    uint32_t seed;
    
    for (uint32_t id = start; id < num_seeds; id += stride) {
        seed = (seed_offsets[id] >> 32);

        // start and end from the seed block_id table
        num_seed_hit = d_index_table[seed];
        if (seed > 0){
            num_seed_hit -= d_index_table[seed-1];
        }

        seed_hit_num[id] = num_seed_hit;
    }
}

__global__
void find_hits (const uint32_t* __restrict__  d_index_table, const uint32_t* __restrict__ d_pos_table, uint64_t*  d_seed_offsets, int seed_size, uint32_t* seed_hit_num, int num_hits, hsp* d_hsp, uint32_t start_seed_index, uint32_t start_hit_index){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int warp_size = warpSize;
    int lane_id = thread_id%warp_size;
    int warp_id = (thread_id-lane_id)/warp_size;

    __shared__ uint32_t start, end;
    __shared__ uint32_t seed;
    __shared__ uint64_t seed_offset;

    __shared__ uint32_t ref_loc[NUM_WARPS];
    __shared__ uint32_t query_loc;
    __shared__ uint32_t seed_hit_prefix;

    if(thread_id == 0){
        seed_offset = d_seed_offsets[block_id+start_seed_index];
        seed = (seed_offset >> 32);
        query_loc = ((seed_offset << 32) >> 32) + seed_size - 1;

        // start and end from the seed block_id table
        end = d_index_table[seed];
        start = 0;
        if (seed > 0){
            start = d_index_table[seed-1];
        }
        seed_hit_prefix = seed_hit_num[block_id+start_seed_index]; 
    }
    __syncthreads();

    for (int id1 = start; id1 < end; id1 += NUM_WARPS) {
        if(id1+warp_id < end){ 
            if(lane_id == 0){ 
                ref_loc[warp_id]   = d_pos_table[id1+warp_id] + seed_size - 1;
                int dram_address = seed_hit_prefix -id1 - warp_id+start-1-start_hit_index;

                d_hsp[dram_address].ref_start = ref_loc[warp_id];
                d_hsp[dram_address].query_start = query_loc; 
                d_hsp[dram_address].len = 0;
                d_hsp[dram_address].score = 0;
            }
        }
    }
}

__global__
void find_anchors (const char* __restrict__  d_ref_seq, const char* __restrict__  d_query_seq, int *d_sub_mat, int xdrop, int hspthresh, uint32_t* d_done, uint32_t ref_len, uint32_t query_len, int seed_size, uint32_t* seed_hit_num, int num_hits, hsp* d_hsp, bool noentropy){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int num_blocks = gridDim.x;
    int warp_size = warpSize;
    int lane_id = thread_id%warp_size;
    int warp_id = (thread_id-lane_id)/warp_size;

    __shared__ uint32_t ref_loc[NUM_WARPS];
    __shared__ uint32_t query_loc[NUM_WARPS];
    __shared__ int total_score[NUM_WARPS];
    __shared__ int prev_score[NUM_WARPS];
    __shared__ int prev_max_score[NUM_WARPS];
    __shared__ uint32_t prev_max_pos[NUM_WARPS];
    __shared__ bool edge_found[NUM_WARPS]; 
    __shared__ bool xdrop_found[NUM_WARPS]; 
    __shared__ uint32_t left_extent[NUM_WARPS];
    __shared__ uint32_t extent[NUM_WARPS];
    __shared__ uint32_t tile[NUM_WARPS];
    __shared__ float entropy[NUM_WARPS];

    int thread_score;
    int max_thread_score;
    uint32_t max_pos;
    uint32_t temp_pos;
    bool xdrop_done;
    int temp;
    short count[4];
    char r_chr;
    char q_chr;
    uint32_t ref_pos;
    uint32_t query_pos;
    uint32_t pos_offset;

    __shared__ int sub_mat[NUC2];

    if(thread_id < NUC2){
        sub_mat[thread_id] = d_sub_mat[thread_id];
    }
    __syncthreads();

    for(int hid0 = block_id*NUM_WARPS; hid0 < num_hits; hid0 += NUM_WARPS*num_blocks){ 
        int hid = hid0 + warp_id; 

        if(hid < num_hits){
            if(lane_id == 0){
                ref_loc[warp_id] = d_hsp[hid].ref_start;
                query_loc[warp_id] = d_hsp[hid].query_start;
                total_score[warp_id] = 0; 
            }
        }
        else{
            if(lane_id == 0){

                ref_loc[warp_id] = d_hsp[hid0].ref_start;
                query_loc[warp_id] = d_hsp[hid0].query_start;
                total_score[warp_id] = 0; 
            }
        }
        __syncwarp();

        //////////////////////////////////////////////////////////////////

        if(lane_id ==0){
            tile[warp_id] = 0;
            xdrop_found[warp_id] = false;
            edge_found[warp_id] = false;
            entropy[warp_id] = 1.0f;
            prev_score[warp_id] = 0;
            prev_max_score[warp_id] = 0;
            prev_max_pos[warp_id] = 0;
            extent[warp_id] = 0;
        }

        count[0] = 0;
        count[1] = 0;
        count[2] = 0;
        count[3] = 0;
        max_pos = 0;

        __syncwarp();

        while(!xdrop_found[warp_id] && !edge_found[warp_id]){
            pos_offset = lane_id + tile[warp_id];
            ref_pos   = ref_loc[warp_id] + pos_offset;
            query_pos = query_loc[warp_id] + pos_offset;
            thread_score = 0;

            if(ref_pos < ref_len && query_pos < query_len){
                r_chr = d_ref_seq[ref_pos];
                q_chr = d_query_seq[query_pos];
                thread_score = sub_mat[r_chr*NUC+q_chr];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                if(lane_id >= offset){
                    thread_score += temp;
                }
            }

            thread_score += prev_score[warp_id];
            if(thread_score > prev_max_score[warp_id]){
                max_thread_score = thread_score;
                max_pos = pos_offset;
            }
            else{
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }

            __syncwarp();
#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }

            xdrop_done = ((max_thread_score-thread_score) > xdrop);
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                xdrop_done |= __shfl_up_sync(0xFFFFFFFF, xdrop_done, offset);
            }

            if(lane_id == warp_size-1){
                if(xdrop_done){
                    total_score[warp_id]+=max_thread_score;
                    xdrop_found[warp_id] = true;
                    extent[warp_id] = max_pos;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else if(ref_pos >= ref_len || query_pos >= query_len){
                    total_score[warp_id] += max_thread_score;
                    edge_found[warp_id] = true;
                    extent[warp_id] = max_pos;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else{
                    prev_score[warp_id] = thread_score;
                    prev_max_score[warp_id] = max_thread_score;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id]+=warp_size;
                }
            }
            __syncwarp();

            if(pos_offset <=  tile[warp_id])
                if(r_chr == q_chr)
                    count[r_chr]++;
            __syncwarp();
        }

        __syncwarp();

        ////////////////////////////////////////////////////////////////

        if(lane_id ==0){
            tile[warp_id] = 0;
            xdrop_found[warp_id] = false;
            edge_found[warp_id] = false;
            prev_score[warp_id] = 0;
            prev_max_score[warp_id] = 0;
            prev_max_pos[warp_id] = 0;
            left_extent[warp_id] = 0;
        }
        max_pos = 0;
        __syncwarp();

        while(!xdrop_found[warp_id] && !edge_found[warp_id]){
            pos_offset = lane_id+1+tile[warp_id];
            thread_score = 0;

            if(ref_loc[warp_id] >= pos_offset  && query_loc[warp_id] >= pos_offset){
                ref_pos   = ref_loc[warp_id] - pos_offset;
                query_pos = query_loc[warp_id] - pos_offset;
                r_chr = d_ref_seq[ref_pos];
                q_chr = d_query_seq[query_pos];
                thread_score = sub_mat[r_chr*NUC+q_chr];

            }

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                if(lane_id >= offset){
                    thread_score += temp;
                }
            }

            thread_score += prev_score[warp_id];
            if(thread_score > prev_max_score[warp_id]){
                max_thread_score = thread_score;
                max_pos = pos_offset;
            }
            else{
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }

            xdrop_done = ((max_thread_score-thread_score) > xdrop);
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                xdrop_done |= __shfl_up_sync(0xFFFFFFFF, xdrop_done, offset);
            }

            if(lane_id == warp_size-1){
                if(xdrop_done){
                    total_score[warp_id]+=max_thread_score;
                    xdrop_found[warp_id] = true;
                    left_extent[warp_id] = max_pos;
                    extent[warp_id] += left_extent[warp_id];
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else if(ref_loc[warp_id] < pos_offset || query_loc[warp_id] < pos_offset){
                    total_score[warp_id]+=max_thread_score;
                    edge_found[warp_id] = true;
                    left_extent[warp_id] = max_pos;
                    extent[warp_id] += left_extent[warp_id];
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else{
                    prev_score[warp_id] = thread_score;
                    prev_max_score[warp_id] = max_thread_score;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id]+=warp_size;
                }
            }
            __syncwarp();

            if(pos_offset <=  tile[warp_id])
                if(r_chr == q_chr)
                    count[r_chr]++;

            __syncwarp();
        }

        //////////////////////////////////////////////////////////////////

        if(total_score[warp_id] >= hspthresh && total_score[warp_id] <= 3*hspthresh && !noentropy){
            for(int i = 0; i < 4; i++){
#pragma unroll
                for (int offset = 1; offset < warp_size; offset = offset << 1){
                    count[i] += __shfl_up_sync(0xFFFFFFFF, count[i], offset);
                }
            }
            __syncwarp();

            if(lane_id == warp_size-1 && ((count[0]+count[1]+count[2]+count[3]) >= 20)){

                entropy[warp_id] = 0.f;
                for(int i = 0; i < 4; i++){
                    entropy[warp_id] += ((float) count[i])/((float) extent[warp_id]) * ((count[i] != 0) ? log(((double) count[i]) / ((double) extent[warp_id])): 0.f); 
                }
                entropy[warp_id] = -entropy[warp_id]/log(4.0f);
            }
        }
        __syncwarp();

        //////////////////////////////////////////////////////////////////

        if(hid < num_hits){
            if(lane_id == 0){

                if( ((int) (((float) total_score[warp_id])  * entropy[warp_id])) >= hspthresh){
                    d_hsp[hid].ref_start = ref_loc[warp_id] - left_extent[warp_id];
                    d_hsp[hid].query_start = query_loc[warp_id] - left_extent[warp_id];
                    d_hsp[hid].len = extent[warp_id];
                    d_hsp[hid].score = total_score[warp_id];
                    d_done[hid] = 1;
                }
                else{
                    d_hsp[hid].ref_start = ref_loc[warp_id];
                    d_hsp[hid].query_start = query_loc[warp_id];
                    d_hsp[hid].len = 0;
                    d_hsp[hid].score = 0;
                    d_done[hid] = 0;
                }
            }
        }
        __syncwarp();
    }
}

std::vector<hsp> SeedAndFilter (std::vector<uint64_t> seed_offset_vector, bool rev, uint32_t buffer, uint32_t seed_size, int xdrop, int hspthresh, bool noentropy, bool nounique){

    hipError_t err;

    uint32_t num_hits = 0;
    uint32_t total_anchors = 0;

    uint32_t num_seeds = seed_offset_vector.size();
    assert(num_seeds <= MAX_SEEDS);

    uint64_t* tmp_offset = (uint64_t*) malloc(num_seeds*sizeof(uint64_t));
    for (uint32_t i = 0; i < num_seeds; i++) {
        tmp_offset[i] = seed_offset_vector[i];
    }

    int g;
    std::unique_lock<std::mutex> locker(mu);
    if (available_gpus.empty()) {
        cv.wait(locker, [](){return !available_gpus.empty();});
    }
    g = available_gpus.back();
    available_gpus.pop_back();
    locker.unlock();

    err = hipSetDevice(g);

    err = hipMemcpy(d_seed_offsets[g], tmp_offset, num_seeds*sizeof(uint64_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed! seed_offsets\n");
        exit(1);
    }

    find_num_hits <<<MAX_BLOCKS, MAX_THREADS>>> (num_seeds, d_index_table[g], d_seed_offsets[g], d_hit_num_array[g]);

    thrust::inclusive_scan(d_hit_num_vec[g].begin(), d_hit_num_vec[g].begin() + num_seeds, d_hit_num_vec[g].begin());

    err = hipMemcpy(&num_hits, d_hit_num_array[g]+num_seeds-1, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed! num_hits\n");
        exit(1);
    }
    
    int iters = num_hits/MAX_HITS+1;

    thrust::device_vector<uint32_t> limit_value (iters); 
    thrust::device_vector<uint32_t> limit_pos (iters); 

    for(int i = 0; i < iters-1; i++)
        limit_value[i] = (i+1)*MAX_HITS;

    limit_value[iters-1] = num_hits+1;

    thrust::lower_bound(d_hit_num_vec[g].begin(), d_hit_num_vec[g].begin()+num_seeds, limit_value.begin(), limit_value.end(), limit_pos.begin()); 

    for(int i = 0; i < iters-1; i++)
        limit_pos[i] --;
    limit_pos[iters-1] = num_seeds-1;

    hsp** h_hsp = (hsp**) malloc(iters*sizeof(hsp*));
    uint32_t* num_anchors = (uint32_t*) calloc(iters, sizeof(uint32_t));
    uint32_t iter_start_index, iter_end_index, iter_start_val, iter_end_val;
    iter_start_index = 0;
    iter_end_index =  limit_pos[0]+1;
    iter_start_val = 0;
    iter_end_val = d_hit_num_vec[g][limit_pos[0]];
    uint32_t iter_num_seeds, iter_num_hits;
    iter_num_seeds = iter_end_index - iter_start_index;
    iter_num_hits = iter_end_val - iter_start_val;

    if(num_hits > 0){

        for(int i = 0; i < iters; i++){

            find_hits <<<iter_num_seeds, BLOCK_SIZE>>> (d_index_table[g], d_pos_table[g], d_seed_offsets[g], seed_size, d_hit_num_array[g], num_hits, d_hsp[g], iter_start_index, iter_start_val);
            if(rev){
                find_anchors <<<1024, BLOCK_SIZE>>> (d_ref_seq[g], d_query_rc_seq[buffer*NUM_DEVICES+g], d_sub_mat[g], xdrop, hspthresh, d_done_array[g], ref_len, query_length[buffer], seed_size, d_hit_num_array[g], num_hits, d_hsp[g], noentropy);
            }
            else{
                find_anchors <<<1024, BLOCK_SIZE>>> (d_ref_seq[g], d_query_seq[buffer*NUM_DEVICES+g], d_sub_mat[g], xdrop, hspthresh, d_done_array[g], ref_len, query_length[buffer], seed_size, d_hit_num_array[g], num_hits, d_hsp[g], noentropy);
            }

            thrust::inclusive_scan(d_done_vec[g].begin(), d_done_vec[g].begin() + iter_num_hits, d_done_vec[g].begin());

            err = hipMemcpy(&num_anchors[i], d_done_array[g]+iter_num_hits-1, sizeof(uint32_t), hipMemcpyDeviceToHost);
            if (err != hipSuccess) {
                fprintf(stderr, "Error: hipMemcpy failed! num_anchors %s\n", hipGetErrorString(err));
                exit(1);
            }

            total_anchors += num_anchors[i];

            if(num_anchors[i] > 0){
                fill_output <<<MAX_BLOCKS, MAX_THREADS>>>(d_done_array[g], d_hsp[g], d_hsp_reduced[g], num_hits);
                if(nounique) {

                    h_hsp[i] = (hsp*) calloc(num_anchors[i], sizeof(hsp));

                    err = hipMemcpy(h_hsp[i], d_hsp_reduced[g], num_anchors[i]*sizeof(hsp), hipMemcpyDeviceToHost);
                    if (err != hipSuccess) {
                        fprintf(stderr, "Error: hipMemcpy failed! hsp with num_anchors= %u\n", num_anchors[i]);
                        exit(1);
                    }
                }

                else {

                    thrust::device_vector<hsp>::iterator result_end = thrust::unique_copy(d_hsp_reduced_vec[g].begin(), d_hsp_reduced_vec[g].begin()+num_anchors[i], d_hsp_vec[g].begin(),  hspEqual());
                    num_anchors[i] = thrust::distance(d_hsp_vec[g].begin(), result_end), num_anchors[i];

                    h_hsp[i] = (hsp*) calloc(num_anchors[i], sizeof(hsp));

                    err = hipMemcpy(h_hsp[i], d_hsp[g], num_anchors[i]*sizeof(hsp), hipMemcpyDeviceToHost);
                    if (err != hipSuccess) {
                        fprintf(stderr, "Error: hipMemcpy failed! hsp with num_anchors= %u\n", num_anchors[i]);
                        exit(1);
                    }
                }
            }

            if(i < iters-1){
                iter_start_index = iter_end_index;
                iter_end_index =  limit_pos[i+1]+1;
                iter_num_seeds = iter_end_index - iter_start_index;

                iter_start_val = iter_end_val;
                iter_end_val = d_hit_num_vec[g][limit_pos[i+1]];
                iter_num_hits = iter_end_val - iter_start_val;
            }
        }
    }

    limit_value.clear();
    limit_pos.clear();

    {
        std::unique_lock<std::mutex> locker(mu);
        available_gpus.push_back(g);
        locker.unlock();
        cv.notify_one();
    }
    std::vector<hsp> gpu_filter_output;

    hsp first_el;
    first_el.len = total_anchors;
    first_el.score = num_hits;
    gpu_filter_output.push_back(first_el);

    if(total_anchors > 0){
        for(int it = 0; it < iters; it++){

            for(int i = 0; i < num_anchors[it]; i++){
                gpu_filter_output.push_back(h_hsp[it][i]);
            }
        }
        free(h_hsp);
    }
    
    free(tmp_offset);
    return gpu_filter_output;
}

size_t InitializeProcessor (int* sub_mat, bool transition, uint32_t WGA_CHUNK){

    size_t ret = 0;
    hipError_t err;
    int nDevices;

    if(transition)
        MAX_SEEDS = 13*WGA_CHUNK;
    else
        MAX_SEEDS = WGA_CHUNK;
    MAX_HITS = MAX_SEEDS * 10;
    MAX_HITS_SIZE = 2*MAX_HITS;

    err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: No GPU device found!\n");
        exit(1);
    }

    NUM_DEVICES = nDevices; 
    fprintf(stderr, "Using %d GPU(s)\n", NUM_DEVICES);

    d_seed_offsets = (uint64_t**) malloc(NUM_DEVICES*sizeof(uint64_t*));
    d_done_array = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_hit_num_array = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_hsp = (hsp**) malloc(NUM_DEVICES*sizeof(hsp*));
    d_hsp_reduced = (hsp**) malloc(NUM_DEVICES*sizeof(hsp*));
    d_sub_mat = (int**) malloc(NUM_DEVICES*sizeof(int*));
    d_done_vec.reserve(NUM_DEVICES);
    d_hit_num_vec.reserve(NUM_DEVICES);
    d_hsp_vec.reserve(NUM_DEVICES);
    d_hsp_reduced_vec.reserve(NUM_DEVICES);
    hsp zeroHsp;
    zeroHsp.ref_start = 0;
    zeroHsp.query_start = 0;
    zeroHsp.len = 0;
    zeroHsp.score = 0;


    for(int g = 0; g < NUM_DEVICES; g++){

        hipSetDevice(g);

        d_done_vec.emplace_back(MAX_HITS_SIZE, 0);
        d_done_array[g] = thrust::raw_pointer_cast(d_done_vec.at(g).data());

        d_hsp_vec.emplace_back(MAX_HITS_SIZE, zeroHsp);
        d_hsp[g] = thrust::raw_pointer_cast(d_hsp_vec.at(g).data());

        d_hsp_reduced_vec.emplace_back(MAX_HITS_SIZE, zeroHsp);
        d_hsp_reduced[g] = thrust::raw_pointer_cast(d_hsp_reduced_vec.at(g).data());

        d_hit_num_vec.emplace_back(MAX_SEEDS, 0);
        d_hit_num_array[g] = thrust::raw_pointer_cast(d_hit_num_vec.at(g).data());

        check_cuda_malloc((void**)&d_seed_offsets[g], MAX_SEEDS*sizeof(uint64_t), "seed_offsets");
        check_cuda_malloc((void**)&d_sub_mat[g], NUC2*sizeof(int), "sub_mat"); 

        err = hipMemcpy(d_sub_mat[g], sub_mat, NUC2*sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Error: hipMemcpy failed! sub_mat\n");
            exit(1);
        }

        available_gpus.push_back(g);
    }
    
    d_query_seq    = (char**) malloc(BUFFER_DEPTH*NUM_DEVICES*sizeof(char*));
    d_query_rc_seq = (char**) malloc(BUFFER_DEPTH*NUM_DEVICES*sizeof(char*));
    d_ref_seq = (char**) malloc(NUM_DEVICES*sizeof(char*));
    d_index_table = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_pos_table = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));

    return ret;
}

void SendRefWriteRequest (size_t start_addr, size_t len){
    hipError_t err;
    ref_len = len;
    
    for(int g = 0; g < NUM_DEVICES; g++){

        hipSetDevice(g);
        char* d_ref_seq_tmp;
        check_cuda_malloc((void**)&d_ref_seq_tmp, len*sizeof(char), "tmp ref_seq"); 

        err = hipMemcpy(d_ref_seq_tmp, g_DRAM->buffer + start_addr, len*sizeof(char), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Error: hipMemcpy failed! ref_seq\n");
            exit(1);
        }

        check_cuda_malloc((void**)&d_ref_seq[g], len*sizeof(char), "ref_seq"); 

        compress_string <<<MAX_BLOCKS, MAX_THREADS>>> (len, d_ref_seq_tmp, d_ref_seq[g]);

        hipFree(d_ref_seq_tmp);
    }
}

void SendQueryWriteRequest (size_t start_addr, size_t len, uint32_t buffer){
    hipError_t err;
    query_length[buffer] = len;

    for(int g = 0; g < NUM_DEVICES; g++){

        hipSetDevice(g);
        char* d_query_seq_tmp;
        check_cuda_malloc((void**)&d_query_seq_tmp, len*sizeof(char), "tmp query_seq"); 

        err = hipMemcpy(d_query_seq_tmp, g_DRAM->buffer + start_addr, len*sizeof(char), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Error: hipMemcpy failed! query_seq\n");
            exit(1);
        }

        check_cuda_malloc((void**)&d_query_seq[buffer*NUM_DEVICES+g], len*sizeof(char), "query_seq"); 
        check_cuda_malloc((void**)&d_query_rc_seq[buffer*NUM_DEVICES+g], len*sizeof(char), "query_rc_seq"); 

        compress_string_rev_comp <<<MAX_BLOCKS, MAX_THREADS>>> (len, d_query_seq_tmp, d_query_seq[buffer*NUM_DEVICES+g], d_query_rc_seq[buffer*NUM_DEVICES+g]);

        hipFree(d_query_seq_tmp);
    }
}

void SendSeedPosTable (uint32_t* index_table, uint32_t index_table_size, uint32_t* pos_table, uint32_t num_index, uint32_t max_pos_index){
    hipError_t err;

    for(int g = 0; g < NUM_DEVICES; g++){

        hipSetDevice(g);

        check_cuda_malloc((void**)&d_index_table[g], index_table_size*sizeof(uint32_t), "index_table"); 

        err = hipMemcpy(d_index_table[g], index_table, index_table_size*sizeof(uint32_t), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Error: hipMemcpy failed! index_table\n");
            exit(1);
        }

        check_cuda_malloc((void**)&d_pos_table[g], num_index*sizeof(uint32_t), "pos_table"); 

        err = hipMemcpy(d_pos_table[g], pos_table, num_index*sizeof(uint32_t), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Error: hipMemcpy failed! pos_table\n");
            exit(1);
        }
    }
}

void InclusivePrefixScan (uint32_t* data, uint32_t len) {
    int g;
    
    {
        std::unique_lock<std::mutex> locker(mu);
        if (available_gpus.empty()) {
            cv.wait(locker, [](){return !available_gpus.empty();});
        }
        g = available_gpus.back();
        available_gpus.pop_back();
        locker.unlock();

        hipError_t err = hipSetDevice(g);
        if (err != hipSuccess) {
            fprintf(stderr, "Error: hipSetDevice failed!\n");
            exit(1);
        }
    }


    thrust::inclusive_scan(thrust::host, data, data + len, data); 

    {
        std::unique_lock<std::mutex> locker(mu);
        available_gpus.push_back(g);
        locker.unlock();
        cv.notify_one();
    }
}

void clearRef(){

    for(int g = 0; g < NUM_DEVICES; g++){

        hipSetDevice(g);
        hipFree(d_ref_seq[g]);
        hipFree(d_index_table[g]);
        hipFree(d_pos_table[g]);
    }
}

void clearQuery(uint32_t buffer){

    for(int g = 0; g < NUM_DEVICES; g++){

        hipSetDevice(g);
        hipFree(d_query_seq[buffer*NUM_DEVICES+g]);
        hipFree(d_query_rc_seq[buffer*NUM_DEVICES+g]);

    }
}

void ShutdownProcessor(){

    d_done_vec.clear();
    d_hit_num_vec.clear();
    d_hsp_vec.clear();
    d_hsp_reduced_vec.clear();

    hipDeviceReset();
}

InitializeProcessor_ptr g_InitializeProcessor = InitializeProcessor;
SendSeedPosTable_ptr g_SendSeedPosTable = SendSeedPosTable;
SeedAndFilter_ptr g_SeedAndFilter = SeedAndFilter;
SendRefWriteRequest_ptr g_SendRefWriteRequest = SendRefWriteRequest;
SendQueryWriteRequest_ptr g_SendQueryWriteRequest = SendQueryWriteRequest;
InclusivePrefixScan_ptr g_InclusivePrefixScan = InclusivePrefixScan;
ShutdownProcessor_ptr g_ShutdownProcessor = ShutdownProcessor;
clearRef_ptr g_clearRef = clearRef;
clearQuery_ptr g_clearQuery = clearQuery;
